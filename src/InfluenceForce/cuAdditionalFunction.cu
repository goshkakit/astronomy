
#include <hip/hip_runtime.h>
//==============================================================================//
// �������������� ������ � ������� ��������� �������� � ������
// mat*mat
//==============================================================================//
__device__ void kernalmatMul( double *in1, double *in2, double *out )
{
	for( int j = 0; j < 3; j++ )
	{
		for( int i = 0; i < 3; i++ )
		{
			out[j*3+i] = 0;
			for( int k = 0; k < 3; k++ )
			{
				out[j*3+i] += in1[j*3+k]*in2[k*3 + i];
			}
		}
	}
}
//==============================================================================//
// mat*vec
//==============================================================================//
__device__ void kernalmatVecMul( double *inMat, double *inVec, double *outVec )
{
	for( int j = 0; j < 3; j++ )
	{
		outVec[j] = 0;
		for( int k = 0; k < 3; k++ )
		{
			outVec[j] += inMat[j*3+k]*inVec[k];
		}
	}
}
//==============================================================================//
// mat*pos and velocity
//==============================================================================//
__device__ void kernalmatVecMul_V6( double *inMat, double *inVec, double *outVec )
{
	for( int j = 0; j < 3; j++ )
	{
		outVec[j] = 0;
		for( int k = 0; k < 3; k++ )
		{
			outVec[j] += inMat[j*3+k]*inVec[k];
		}
	}

	for( int j = 0; j < 3; j++ )
	{
		outVec[j+3] = 0;
		for( int k = 0; k < 3; k++ )
		{
			outVec[j+3] += inMat[j*3+k]*inVec[k+3];
		}
	}
}
//==============================================================================//
// ���������������� ������
//==============================================================================//
__device__ void kernaltranspose( double *A, double *B )
{
	for( int j = 0; j < 3; j++ )
	{
		for( int i = 0; i < 3; i++ )
		{
			B[i*3+j] = A[j*3+i];
		}
	}
}

//==============================================================================//
// �������������� ������� �������������� ������ � ����������
//==============================================================================//
__device__ double kernalDMOD( double X, double Y )
{
	int s = (int)(X/Y);
	double res = X - ((double)s)*Y;

	return res;
}

__device__ double kernalDDIM( double X, double Y  )
{
	double res = X-Y;
	if( res < 0 )
		res = 0;

	return res;
}

__device__ double kernalDSIGN( double X, double Y )
{
	double sig = 1;
	if( Y < 0 )
		sig = -1;
	if( Y == 0 )
		sig = 0;

	X = sig*X;

	return X;
}
__device__ double kernalDmax( double X, double Y )
{
	if( X > Y )
		return X;
	else
		return Y;
}
__device__ double kernalDmin( double X, double Y )
{
	if( X < Y )
		return X;
	else
		return Y;
}
//==============================================================================//