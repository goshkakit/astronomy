#include "hip/hip_runtime.h"
//==============================================================================//
// Andrianov N.G.
// opbit predict 
// module find Influence Force
// Earth Atmosphere
// GPU version
//==============================================================================//
#include <math.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

//==============================================================================//
// ��������� �������������� �������
//==============================================================================//
__device__ int kernalAnint( double X )
{
	double res;
	int iX = (int)X;
	double delta = X - (double)iX;

	if( delta > 0.5 )
		res = iX + 1;
	else
		res = iX;

	return iX;
};
__device__ double kernalAmax( double X, double Y )
{
	if( X > Y )
		return X;
	else
		return Y;
}
__device__ double kernalAmin( double X, double Y )
{
	if( X < Y )
		return X;
	else
		return Y;
}
__device__ double kernalAmod( double X, double Y )
{
	int s = (int)(X/Y);
	double res = X - ((double)s)*Y;

	return res;
}
//==============================================================================//
// ������� ������ ��������� ������� ����� ��������� �������� �����.
//==============================================================================//
__device__  double kernalRoa2004_2( double time, double *x, double ajd0, double delt0 )
{
	double f0t[7] = {  75.0, 100.0, 125.0, 150.0, 175.0, 200.0, 200.0 };
	double Re = 6378.136;
	double om = 6.300388008;
	double alpha = 0.0033528037;
	double b1900 = 2415020.31352;
	double  btau = 365.2422;
	double  ro0 = 1.58868E-8;
	double   an0 =  .20580000E+01;
	double   an1 =  .58870000E-02;
	double  an2 = -.40120000E-05;

	double saem[4];
	saem[0] = 100.0;
	saem[1] = 100.0;
	saem[2] = 100.0;
	saem[3] = 3.0;

	double Rmod, sinF, h, F107, F81, aKp, F0;
	double e0, e1, e2, e3, e4, e5, e6, e7, e8;
	double a0, a1, a2, a3, a4, a5, a6;
	double al0, al1, al2, al3, al4;
	double c0, c1, c2, c3, c4;
	double d0, d1, d2, d3, d4;
	double b0, b1, b2, b3, b4;
	double fi1, power, aKs0, aKs1, aKs2, aKs3, aKs4, aKss1, aKss4, ron;
	double ct, sz, xss, yss, zss, rr;
	double cosFi1, sinFi1, app, s_app, c_app, xs, ys, zs, rs, cosFi, cos05, ak0, ak1, d, ad, ak2, ak3, ak4;
	double h0, akst1, akst2, ast0, t;
	int iq, iq6 ,ish_a, ish_b, ish_c, ish_d, ish_e, ish_l;

	t = ajd0+(delt0+time)/86.4;
	Rmod = sqrt( x[0]*x[0] + x[1]*x[1] + x[2]*x[2] );
	sinF = x[2]/Rmod;
	h = kernalAmax( Rmod*1.0E+3-Re*(1.0-alpha*sinF*sinF), 0.0);
		
	double roa2004 = 0;

	if (h >= 1500.0) 
	{
		roa2004 = 0.0;
		return roa2004;
	}
	else if (h > 120.0) 
	{
		F107 = saem[1];
		F81  = saem[2];
		aKp  = saem[3];
		iq = kernalAmin( kernalAmax( kernalAnint((F81-75.0)/25.0), 0), 7) +1;
			
		F0 = f0t[iq-1]; //!!!

		if( F81 > 200.0)
		{
			iq = 6;
		}
		else if (F81 > 225.0) 
		{
			F0 = 250.0;
			iq = 7;
		}

		iq6 = (iq-1)*6;

		iq6 = iq6 - 1; //!! ��������� �������

		ish_a=0;
		ish_b=0;
		ish_c=0;
		ish_d=0;
		ish_e=0;
		ish_l=0;
		if (h > CUtick[iq6+1]) ish_a=7;
		if (h > CUtick[iq6+2]) ish_b=7;
		if (h > CUtick[iq6+3]) ish_c=7;
		if (h > CUtick[iq6+4]) ish_d=7;
		if (h > CUtick[iq6+5]) ish_e=7;
		if (h > CUtick[iq6+6]) ish_l=7;

		// ------------------
		iq = iq - 1; //!! ��������� �������
		// a - coeffs
		a0= CUaa0[ish_a+iq];
		a1= CUaa1[ish_a+iq];
		a2= CUaa2[ish_a+iq];
		a3= CUaa3[ish_a+iq];
		a4= CUaa4[ish_a+iq];
		a5= CUaa5[ish_a+iq];
		a6= CUaa6[ish_a+iq];
		// al - coeffs
		al0= CUaal0[ish_l+iq];
		al1= CUaal1[ish_l+iq];
		al2= CUaal2[ish_l+iq];
		al3= CUaal3[ish_l+iq];
		al4= CUaal4[ish_l+iq];
		// c - coeffs
		c0= CUcc0[ish_c+iq];
		c1= CUcc1[ish_c+iq];
		c2= CUcc2[ish_c+iq];
		c3= CUcc3[ish_c+iq];
		c4= CUcc4[ish_c+iq];
		// d - coeffs
		d0= CUdd0[ish_d+iq];
		d1= CUdd1[ish_d+iq];
		d2= CUdd2[ish_d+iq];
		d3= CUdd3[ish_d+iq];
		d4= CUdd4[ish_d+iq];
		// b - coeffs
		b0= CUbb0[ish_b+iq];
		b1= CUbb1[ish_b+iq];
		b2= CUbb2[ish_b+iq];
		b3= CUbb3[ish_b+iq];
		b4= CUbb4[ish_b+iq];
		// e - coeffs
		e0= CUee0[ish_e+iq];
		e1= CUee1[ish_e+iq];
		e2= CUee2[ish_e+iq];
		e3= CUee3[ish_e+iq];
		e4= CUee4[ish_e+iq];
		e5= CUee5[iq];
		e6= CUee6[iq];
		e7= CUee7[iq];
		e8= CUee8[iq];
		// ------------------

		fi1 = CUff1[iq];

		power = a0+h*(a1+h*(a2+h*(a3+h*(a4+h*(a5+h*a6)))));
		aKs0 = al0+h*(al1+h*(al2+h*(al3+h*al4)));
		aKs1 = c0+h*(c1+h*(c2+h*(c3+h*c4)));
		aKss1 = an0+h*(an1+h*an2);
		aKs2 = d0+h*(d1+h*(d2+h*(d3+h*d4)));
		aKs3 = b0+h*(b1+h*(b2+h*(b3+h*b4)));
		aKs4 = e0+h*(e1+h*(e2+h*(e3+h*e4)));
		aKss4 = e5+aKp*(e6+aKp*(e7+aKp*e8));
		ron = ro0*exp(power);

		ct  = (t-2415020.0)/36525.0;
		sz  = 628.33195099*ct+1.739935890;
		xss = -cos(sz);
		sinF = -sin(sz);
		yss = 0.91747*sinF;
		zss = 0.397805*sinF;
		rr = sqrt( x[0]*x[0] + x[1]*x[1] + x[2]*x[2] );

		cosFi1 = cos(fi1);
		sinFi1 = sin(fi1);
		app = sz+( kernalAmod(t, 1.0)-0.5)*om;
		s_app = sin(app);
		c_app = cos(app);
		xs = xss*c_app + yss*s_app;
		ys = yss*c_app - xss*s_app;
		zs = zss;
		rs = sqrt(xs*xs+ys*ys+zs*zs);
		cosFi = (x[0]*(xs*cosFi1-ys*sinFi1)+ x[1]*(ys*cosFi1+xs*sinFi1)+ x[2]*zs)/(rs*Rmod);
		cos05 = sqrt((1.0+cosFi)/2.0);
		ak0 = 1.0 + aKs0*(F81-F0)/F0;

		//aK1 = aKs1*(cos05**aKss1); //!!!
		ak1 = aKs1*pow(cos05, aKss1);

		d = kernalAmod(t-b1900, btau);

		ad = -2.53418E-2 + d*(-2.44075E-3+d*(3.08389E-6 +d*( 
				2.90115E-6 + d*(-4.99606E-8+d*(3.36327E-10+d*( 
			-1.0966E-12 + d*(1.73227E-15+d*(-1.06271E-18))))))));
		ak2 = ad*aKs2;
		ak3 = aKs3*(F107-F81)/(F81+abs(F107-F81));
		ak4 = aKs4*aKss4;
		roa2004 = ron*ak0*(1.0 + ak1 + ak2 + ak3 + ak4);
		return roa2004;
	}
	else if ((h > 0.0) && (h < 20.0)) 
	{
		h0=0.0;
		ast0=1.228;
		akst1=-9.0764E-2;
		akst2=-2.0452E-3;
	}
	else if ((h >= 20.0)&&(h < 60.0))
	{
		h0=20.0;
		ast0=9.013E-2;
		akst1=-0.16739;
		akst2=6.2669E-4;
	}
	else if ((h >= 60.0)&&(h < 100.0)) 
	{
		h0=60.0;
		ast0=3.104E-4;
		akst1=-0.137;
		akst2=-7.8653E-4;
	}
	else if ((h >= 100.0)&&(h < 120.0)) 
	{
		h0=100.0;
		ast0=3.66E-7;
		akst1=-0.18553;
		akst2=1.5397E-3;
	}
	power = (h-h0)*(akst1+akst2*(h-h0));
	roa2004 = ast0*exp(power);
	return roa2004;
}

//==============================================================================//
// ��������� ������ ����������, ���������� �������������� ���������.
//==============================================================================//
__device__ void kernalAtm_drag( double *x, double t, double *f, double sigma_up, double ajd0, double delt0 )
{
	double v;
	v = sqrt( x[3]*x[3] + x[4]*x[4] + x[5]*x[5] );

	double rc = kernalRoa2004_2( t, x, ajd0, delt0 );
	double coeff =  rc*sigma_up*v*1.0E+6;

	f[0] = -x[3]*coeff;
	f[1] = -x[4]*coeff;
	f[2] = -x[5]*coeff;
}
//==============================================================================//
