#include "hip/hip_runtime.h"
//==============================================================================//
// Andrianov N.G.
// opbit predict 
// module find Influence Force
// Nutation Earth GPU
//==============================================================================//
#include <math.h>
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

#include "cuAdditionalFunction.cu"
//==============================================================================//
// ������ �������� ������� ���������
//==============================================================================//
__device__ double kernalE2000( double E1, double E2)
{
	double E0, THJ, S, T1, T2, DT, T12, T13, DT2, DT3;
	E0 = 2451545.0;
	THJ = 36525.0;
	S = 206264.806;                                                       
	T1=(E1-E0)/THJ;                                     
	T2=(E2-E0)/THJ;                                     
	DT=T2-T1;                                           
	T12=T1*T1;
	T13=T12*T1;                                          
	DT2=DT*DT;                                          
	DT3=DT*DT2;
	//NAKLON  EkLIPTIKI k EKBATORY  
	double rE2000 =  ( 84381.4480 - 46.81500*T1 - 0.000590*T12 + 0.0018130*T13)
		+(-46.81500 - 0.001170*T1 + 0.0054390*T12)*DT
		+(-0.000590 + 0.0054390*T1)*DT2+0.0018130*DT3;
	rE2000=rE2000/S; 
	return rE2000;                                            
} 
//==============================================================================//
//
//==============================================================================//
__device__ void kernalFA2000( double AED, double *FA )
{
	double E0, THJ, PII, PI, T, DT2, DT3, A, B, C, P, R;
	E0 = 2451545.0;
	THJ = 36525.0;

	PII=6.283185307179586;
	PI=3.141592653589793;
	T=(AED-E0)/THJ;
	DT2=T*T;
	DT3=T*DT2;

	A=PI*2650.0+3.470890870;
	FA[0]=2.3555483930+A*T+0.1517952E-3*DT2+0.3103E-6*DT3;

	B=PI*198.0+6.26661061;
	FA[1]=6.24003594+B*T-0.27974E-5*DT2-0.582E-7*DT3;

	C=PI*2684.0+1.431476080;
	FA[2]=1.627901930+C*T-0.642717E-4*DT2+0.533E-7*DT3;

	P=PI*2472.0+5.36010650;
	FA[3]=5.198469510+P*T-0.34085E-4*DT2+0.921E-7*DT3;

	R=PI*10.0+2.34111940;
	FA[4]=2.182438620-R*T+0.361429E-4*DT2+0.388E-7*DT3;

	for( int it = 0; it < 5; it++ )
		FA[it]= kernalDMOD( FA[it], PII ) + PII*kernalDDIM(- kernalDSIGN( 1.0, FA[it] ), 0.0 );

}
//==============================================================================//
// ���������� �������� �������
//==============================================================================//
__device__ void kernalN2000( int N, double AJD, double *HYT, double *cuARG, double *cuAMPL )
{
	double FA[5];
	double B[2];

	double EO, THJ, S, T, A;

	EO = 2451545.0;
	THJ = 36525.0;
	S = 206264.806;

	T = (AJD - EO)/THJ;

	kernalFA2000( AJD, FA );

	HYT[0] = 0.0;
	HYT[1] = 0.0;
	int L = 0;
	for( int i=0; i < N; i++ )
	{
		A = ( cuARG[L]*FA[0] + cuARG[L+1]*FA[1] + cuARG[L+2]*FA[2] + cuARG[L+3]*FA[3] + cuARG[L+4]*FA[4] );

		HYT[0] = ( cuAMPL[L+1] + cuAMPL[L+2]*T)*sin(A) + HYT[0];
		HYT[1] = ( cuAMPL[L+3] + cuAMPL[L+4]*T)*cos(A) + HYT[1];
		B[0] = HYT[0]/S;
		B[1] = HYT[1]/S;
		// PRINT 2,(B(K),K=1,2)
		L=L+5;
	}

	// HYTAWIJ B DOLGOTE B PAD.
	HYT[0]=HYT[0]/S;
	// HYTAWIJ B HAKLOHE B PAD.
	HYT[1]=HYT[1]/S;
	// 1    FORMAT (1X,'BFA200',5f18.11)
	// 2    FORmat (1X,2f20.11)
	// 3    format (1x,e10.3)
}
//==============================================================================//
// ����� ������� �������
//==============================================================================//
__device__ void kernalNM2000( double E, double *HUT, double *cuARG, double *cuAMPL )
{
	double A, Epsi, COS1, COS2, SIN1, SIN2, SIN3;
	int N;
	double HYT[2];

	//INTERFACE 
	//      FUNCTION E2000(E1,E2)
	//      REAL(KIND=8) :: E1
	//      REAL(KIND=8) :: E2
	//      REAL(KIND=8) :: E2000
	//      END FUNCTION E2000
	//   END INTERFACE 

	N = 106;      
	A = kernalE2000( E, E );
	kernalN2000( N, E, HYT, cuARG, cuAMPL );

	Epsi = A + HYT[1];
	COS1 = cos( A );
	COS2 = cos( Epsi );
	SIN1 = sin( HYT[0] );
	SIN2 = sin( A );
	SIN3 = sin( Epsi );

	//MATPITSA HYTATSII
	HUT[0] = cos( HYT[0] );
	HUT[1] = -SIN1*COS1;
	HUT[2] = -SIN1*SIN2;
	HUT[3] = SIN1*COS2;
	HUT[4] = HUT[0]*COS2*COS1+SIN3*SIN2;
	HUT[5] = HUT[0]*COS2*SIN2-SIN3*COS1;
	HUT[6] = SIN1*SIN3;
	HUT[7] = HUT[0]*SIN3*COS1-COS2*SIN2;
	HUT[8] = HUT[0]*SIN3*SIN2+COS2*COS1;
}
//==============================================================================//
