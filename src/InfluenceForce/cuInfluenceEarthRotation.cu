#include "hip/hip_runtime.h"
//==============================================================================//
// Andrianov N.G.
// opbit predict 
// module find Influence Force
// Nutation Earth
// GPU version
//==============================================================================//

#include "hip/hip_runtime.h"
#include ""


#include "cuInfluiencePrecessionEarth.cu"
#include "cuInfluencePoleEarth.cu"
#include "cuInfluenceNutationEarth.cu"
//==============================================================================//
// ��������� �������� ������� ��������� �� EME2000 � ����������� ��� �������� ��.
// ���������, ����������� ������� ������ ��������� � �������, ��������� � ������
// ���� �� �� ��������� �� C, �� ��� ��
//==============================================================================//
__device__ void kernalstate_to_itrf( double t, double *x_2000, double *x_g, double *A_rot )
{
	double omega_earth = 0.072921150850;
	// ���������� ������� ��������
	//double t_epsilon = 1.0E-6;
	//if (abs(t-t_iers_update).GT.t_epsilon) 
	//	iers_update_matrix(t)

	//matVecMul( A_rot, x_2000, x_g );
	kernalmatVecMul_V6( A_rot, x_2000, x_g );

	x_g[3] = x_g[3] + omega_earth*x_g[1];
	x_g[4] = x_g[4] - omega_earth*x_g[0];
};
//==============================================================================//
// ��������� ��������� ������� ��������������� ��������� �������� �����.
//==============================================================================//
__device__ void kernalER_mat ( double t, double d_UT1, double *A_rotat, double ajd0, double delt0, double *cuARG, double *cuAMPL )
{
	double E0 = 2451545.0;
	double THJ = 36525.0;
	double pi2 = 6.2831853071795860;
	double hyt[2];
	double ID, JD, t100, s, a, q, z, dUT1, D;

	dUT1=d_UT1/86400.0;
	ID = ajd0 + (delt0+t)/86.40 - E0 - dUT1;

	double tmpv = (delt0+t)/86.40;

	D = kernalDMOD(ajd0, 1.0) + kernalDMOD(tmpv, 1.0)-dUT1 + 0.50;
	t100 = ID/THJ;

	s = 24110.548410+ID*236.5553679080+D*86400.0 + t100*t100*(0.0931040-t100*6.2E-6);

	JD = ajd0+(delt0+t)/86.40 + dUT1;

	a = kernalE2000(JD, JD);
	kernalN2000( 106, JD, hyt, cuARG, cuAMPL );

	q = s/86400.0+hyt[0]*cos(a)/pi2;

	a = int(q);
	z = (q-a)*pi2;

	for( int it = 0; it < 9; it++ )
		A_rotat[it] = 0.0;

	A_rotat[0] = cos(z);
	A_rotat[1] = sin(z);
	A_rotat[3] = -sin(z);
	A_rotat[4] = cos(z);
	A_rotat[8] = 1.0;
}
//==============================================================================//
// ��������� ��������� ������� �������� �� ITRF � ICRF. ����� �������, 
// ������� �������� �� ����������� ����������� ������� ���������, 
// ��������������� �� ������ ������� t, � ������� EME2000.
//==============================================================================//
__device__ void kernaliers_mat( double t, double *A, double ajd0, double delt0, double *cuARG, double *cuAMPL, double* cufinals_tab, int cufinals_n )
{
	double jd2k = 2451545.0;
	double xyt[3];
	double A_pole[9];
	double A_rotat[9];
	double A_prc[9];
	double A_nut[9];
	double jd;

	jd = ajd0+(delt0+t)/86.40;

	// ���������
	kernalPM2000(jd2k, jd, A_prc);
	// �������
	kernalNM2000(jd, A_nut, cuARG, cuAMPL );
	// �������� �������
	kernalget_xyt( t, xyt, ajd0, delt0, cufinals_tab, cufinals_n );
	kernalPM_mat(xyt[0], xyt[1], A_pole);
	// �������� ��������
	kernalER_mat(t, xyt[2], A_rotat, ajd0, delt0, cuARG, cuAMPL );

	// �������������� ������� ��������
	double T1[9], T2[9];
	kernalmatMul( A_pole, A_rotat, T1 );
	kernalmatMul( T1 ,A_nut, T2 );
	kernalmatMul( T2 ,A_prc, A );
}

//==============================================================================//
// ��������� ���������� �������� �������� ������� �������� �� �������-
// ���� ����������� �� � EME2000. ��� ���� ������� �������� �����������
// � ��������� ���������� A_gr.
//==============================================================================//
//__device__ void kernaliers_update_matrix( double t, double *A_rot, double ajd0, double delt0 )
//{
//	kernaliers_mat(t, A_rot, ajd0, delt0 );
//	double t_iers_update = t;
//}
//==============================================================================//