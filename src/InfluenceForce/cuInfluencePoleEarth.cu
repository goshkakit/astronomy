//==============================================================================//
// Andrianov N.G.
// opbit predict 
// module find Influence Force
// Motion Pole Earth
// GPU version
//==============================================================================//
#include <stdio.h>
#include <math.h>

#include "hip/hip_runtime.h"


//==============================================================================//
// ��������� ��������� �������� ��������� ������ ����� � ������ �� ���-
// ���� � ������ t.
//==============================================================================//
__device__ void kernalget_xyt ( double t, double *xyt, double ajd0, double delt0, double* cufinals_tab, int cufinals_n )
{
	double jd0000 = 2400000.0;
	double xyt4[4];
	double jd, jd00, res;
	int k, k5;

	xyt[0] = 0.0;
	xyt[1] = 0.0;
	xyt[2] = 0.0;

	jd   = ajd0+(delt0+t)/86.40;
	jd00 = cufinals_tab[1] + jd0000;

	if ((jd < 23.0E+5) || (jd > 25.0E+5)) 
	{
		printf( "Module IERS -> get_xyt : unexpected Julian date. Return\n" );
		return;
	}
	k = int(jd-jd00)+1;
	if (k <= 0) 
	{
		printf( "jd = %f jd00 = %f\n", jd, jd00 );
		printf( "Module IERS -> get_xtr : Time is less than first record of finals.dat table. Return\n" );
	}
	if (k > cufinals_n-2) 
	{
		k = cufinals_n-2;
	}
	else if (k < 1) 
	{
		k = 1;
	}
	res = jd - ( jd00 + (double)(k-1) );
	//res = jd - ( jd00 + (double)(k) );
	k5 = (k-1)*5;
	//k5 = (k)*5;

	//xyt4 = finals_tab(k5+2:k5+5)*(1.d0-res)+finals_tab(k5+7:k5+10)*res
	xyt4[0] = cufinals_tab[k5+2]*(1.0-res) + cufinals_tab[k5+7]*res;
	xyt4[1] = cufinals_tab[k5+3]*(1.0-res) + cufinals_tab[k5+8]*res;
	xyt4[2] = cufinals_tab[k5+4]*(1.0-res) + cufinals_tab[k5+9]*res;
	xyt4[3] = cufinals_tab[k5+5]*(1.0-res) + cufinals_tab[k5+10]*res;

	//xyt = xyt4([1, 2, 4])
	xyt[0] = xyt4[0];
	xyt[1] = xyt4[1];
	xyt[2] = xyt4[3];
}
//==============================================================================//
// ��������� ������� ������� �������� ��������������� �������� �������
//==============================================================================//
__device__ void kernalPM_mat ( double x, double y, double *A_pole )
{
	double a, qx, qy, gamma, x2y2;
	qx   = x*x;
	qy   = y*y;
	x2y2 = qx+qy;
	if( (x2y2) < 1.0E-18)
	{
		// �������� ������� ������� ��� ����� �������:
		// ������� ������� - ���������
		for( int it = 0; it < 9; it++ )
			A_pole[it] = 0.0;

		A_pole[0] = 1.0;
		A_pole[4] = 1.0;
		A_pole[8] = 1.0;
	}
	else
	{
		// ������� �������, ��������������� �������� �������
		a     = sqrt(1.0-x2y2);
		gamma = -x*y*(a-1.0)/(x2y2);
		A_pole[0] = (qy+qx*a)/(x2y2);
		A_pole[4] = (qx+qy*a)/(x2y2);
		A_pole[1] = gamma;
		A_pole[3] = gamma;
		A_pole[8] = a;

		A_pole[2] = x;
		A_pole[6] = -x;
		A_pole[5] = -y;
		A_pole[7] = y;
	}
}
//==============================================================================//