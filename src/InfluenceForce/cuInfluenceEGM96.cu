//==============================================================================//
// Andrianov N.G.
// opbit predict 
// module find Influence Force
// find force from Earth harmonics
// gpu version
//==============================================================================//
#include <math.h>

#include "hip/hip_runtime.h"



//==============================================================================//
// ��������� ������� ���������� �� ��������������� ��������������� ���� �����.
// ������ x ���������� � ��������� � ������ ��.
// Re - earht equatorial radius
//==============================================================================//
__device__ void kernalGetF_Harm_egm96( double *x_b, int n_harm, double *f_harm, double *CEGM96 )
{
	// ���������
	double muE = 398.60044150;
	double Re  = 6.3781360;

	double eps = 2.0E-12;
	int n, m, n1, n21, pos_mm, pos_nm1, pos_nm;
	double x[3];
	double ro, rv, w1, w2, w3, w4, dw2, dw3, dw;
	double cosL, sinL, cosmL, sinmL, cosm1L, sinm1L;
	double R, R_n, R_m, tmp, P_nm, P_n1m, P_mm, PR;

	ro = x_b[0]*x_b[0]+x_b[1]*x_b[1];
	rv = sqrt(x_b[2]*x_b[2]+ro);

	x[0] = x_b[0]/rv;
	x[1] = x_b[1]/rv;
	x[2] = x_b[2]/rv;

	f_harm[0] = 0.0;
	f_harm[0] = 0.0;
	f_harm[0] = 0.0;

	ro = sqrt(ro);
	if(ro > eps)
	{
		cosL = x_b[0]/ro;
		sinL = x_b[1]/ro;
	}
	else
	{
		cosL = 1.0;
		sinL = 0.0;
	}

	ro = ro/rv;
	R  = Re/rv;
	w1 = 0.0;
	w2 = 0.0;
	w3 = 0.0;
	w4 = 0.0;
	// �������� cos(ml), sin(ml), cos((m-1)l), sin((m-1)l) ��� ���������� �������� m = 1
	cosm1L = 1.0;
	sinm1L = 0.0;
	cosmL  = cosL;
	sinmL  = sinL;
	// ������� � ������� ������������� egm96(pos_nm) = C_nm, egm96(pos_nm+1) = S_nm
	// �� ����������� S_n0 = 0, ������� �� ������ � ������.

	//############ ������� ��������� ��� egm96,
	//############ ������ ������������� ���������

	pos_mm  = 2;
	pos_nm1 = 1;
	P_mm = 1.0;
	R_m  = R;
	//do m = 1,n_harm 
	for( m = 1; m <=n_harm; m++ )
	{
		// ������� ����, ������ �� m �� 1 �� ����� ��������������� ��������
		// ��������� �������� �������������� ������� ��������
		P_nm   = P_mm;
		P_n1m  = 0.0;
		pos_nm = pos_mm;
		R_n = R_m;
		n1  = m+1;
		dw2 = 0.0;
		dw3 = 0.0;
		// do n = m,n_harm
		for( n = m; n <=n_harm; n++ )
		{
			//! ���������� ����, ������� �� n �� m �� ����� ��������������� ��������
			PR = P_nm*R_n;
			//dw  = PR*( cuegm96[pos_nm]*cosmL + cuegm96[pos_nm+1]*sinmL); 
			dw  = PR*( CEGM96[pos_nm]*cosmL + CEGM96[pos_nm+1]*sinmL);
			w1  = w1 + ((double)(n1))*dw;
			dw2 = dw2 + dw;
			//dw3 = dw3 + PR*(-cuegm96[pos_nm]*sinmL + cuegm96[pos_nm+1]*cosmL);
			//w4  = w4 + PR*( cuegm96[pos_nm1]*cosm1L + cuegm96[pos_nm1+1]*sinm1L);
			dw3 = dw3 + PR*(-CEGM96[pos_nm]*sinmL + CEGM96[pos_nm+1]*cosmL);
			w4  = w4 + PR*( CEGM96[pos_nm1]*cosm1L + CEGM96[pos_nm1+1]*sinm1L);
			R_n = R_n*R;
			tmp = P_n1m;
			P_n1m = P_nm;
			n21 = n+n1;
			P_nm = ( ((double)(n21))*x[2]*P_n1m - ((double)(n+m))*tmp)/((double)(n1-m));
			n1 = n1+1;
			pos_nm1 = pos_nm1 + n21;
			pos_nm = pos_nm + n21;
		}
		w2 = ((double)(m))*dw2+w2;
		w3 = ((double)(m))*dw3+w3;
		R_m = R_m*R;
		cosm1L = cosmL;
		sinm1L = sinmL;
		cosmL = cosm1L*cosL - sinm1L*sinL;
		sinmL = cosm1L*sinL + sinm1L*cosL;
		n21 = m+m+1;
		P_mm = ((double)(n21))*ro*P_mm;
		pos_nm1 = n21 + pos_mm;
		pos_mm = pos_nm1 + 2;
	}
	//! ������������ ������ � m=0
	P_n1m = 1.0;
	P_nm = x[2];
	pos_mm = 1;
	R_n = R;
	w1 = w1*ro;
	//do n = 1,n_harm
	for( n = 1; n <=n_harm; n++ )
	{
		//w1 = w1 + ((double)(n+1))*cuegm96[pos_mm]*R_n*P_nm;
		w1 = w1 + ((double)(n+1))*CEGM96[pos_mm]*R_n*P_nm;
		if (n != n_harm )
		{
			n1 = n+1;
			R_n = R_n*R;
			tmp = P_n1m;
			P_n1m = P_nm;
			n21 = n+n1;
			P_nm = ( ((double)(n21))*x[2]*P_n1m - ((double)(n))*tmp )/n1;
			pos_mm = pos_mm +n21;
		}
	}
	//! ���������� ������������ ���������
	tmp = -muE/rv/rv;
	w2 = w4*ro - w2*x[2];
	f_harm[2] = (w1*x[2] - w2*ro)*tmp;
	w2 = w2*x[2];
	f_harm[0] = (w2*cosL + w3*sinL + w1*x[0])*tmp;
	f_harm[1] = (w2*sinL - w3*cosL + w1*x[1])*tmp;
}

//==============================================================================//
// ��������� ��������� ���������� �����������
//==============================================================================//
__device__ void kernalGetHarmForce( double *x, double *Fharm, double *cuegm96 )
{
	//int n = 75;
	kernalGetF_Harm_egm96( x, 75, Fharm, cuegm96 );
};
//==============================================================================//