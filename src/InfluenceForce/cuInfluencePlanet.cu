//==============================================================================//
// Andrianov N.G.
// opbit predict 
// module find Influence Force
// find force from planet
//==============================================================================//
#include "hip/hip_runtime.h"


//==============================================================================//
// ���������� ������
//==============================================================================//
__device__ void kernalpln_coords( int i, double *pos, double *PLCOORD )
{
	pos[0] = PLCOORD[i*3 + 0];
	pos[1] = PLCOORD[i*3 + 1];
	pos[2] = PLCOORD[i*3 + 2];
}
//==============================================================================//
// �������������� ��������� ������
//==============================================================================//
//double InfluenceForce::mu_plan( int i )
//{
//	double mu_plan[11] = {
//		22.03208047245,		// ��������
//		324.8587656142,		// ������
//		398.6004415,		// �����
//		42.828287,			// ����
//		126712.59708,		// ������
//		37939.51971,		// ������
//		5780.158533417,		// ����
//		6871.307771094,		// ������
//		1.02086,			// ������
//		4.90279914,			// ����
//		132712439.935 };	// ������
//
//		return mu_plan[i];
//}
//==============================================================================//
// ��������� �� ���������� �� ������
// ��������
// ������
// �����
// ����
// ������
// ������
// ����
// ������
// ������
// ����
// ������
//==============================================================================//
__device__ int kernalpln_flag( int i )
{
	//int FLFlag[11] = { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 };
	return 1;
}
//==============================================================================//
// ���������� ������ �� �������� ��������
// GPU
//==============================================================================//
__device__ void kernalcheb2( int mode, double x, int degree, double *coeff, double *w, double *dw, double *ddw )
{
	double bk, bk1, bk2, dbk, dbk1, dbk2, ddbk, ddbk1, ddbk2;

	switch( mode ) 
	{
	case 1:
		bk1 = 0;
		bk2 = 0;
		//do i = degree,1,-1
		for( int i = degree; i > 0; i-- )
		{
			bk = coeff[i] - bk2 + 2.0*bk1*x;
			bk2 = bk1;
			bk1 = bk;
		}
		*w = coeff[0] - bk2 + bk1*x;
		//*dw = 0.0;
		//*ddw = 0.0;
		break; 

	case 2:
		bk1 = 0.0;
		bk2 = 0.0;
		dbk1 = 0.0;
		dbk2 = 0.0;
		//do i = degree,1,-1
		for( int i = degree; i > 0; i-- )
		{
			bk = coeff[i]-bk2 + 2.0*bk1*x;
			bk2 = bk1;
			bk1 = bk;
			dbk = 2.0*(dbk1*x + bk2) - dbk2;
			dbk2 = dbk1;
			dbk1 = dbk;
		}
		*w = coeff[0] - bk2 + bk1*x;
		*dw = bk1 + dbk1*x - dbk2;
		//*ddw = 0.0;
		break; 

	case 3:
		bk1 = 0.0;
		bk2 = 0.0;
		dbk1 = 0.0;
		dbk2 = 0.0;
		ddbk1 = 0.0;
		ddbk2 = 0.0;
		//do i = degree,1,-1
		for( int i = degree; i > 0; i-- )
		{
			bk = coeff[i] - bk2 + 2.0*bk1*x;
			bk2 = bk1;
			bk1 = bk;
			dbk = 2.0*(dbk1*x + bk2) - dbk2;
			dbk2 = dbk1;
			dbk1 = dbk;
			ddbk = 2.0*(ddbk1*x + 2.0*dbk2) - ddbk2;

			ddbk2 = ddbk1;
			ddbk1 = ddbk;
		}
		*w = coeff[0] - bk2 + bk1*x;
		*dw = bk1 + dbk1*x - dbk2;
		*ddw = 2.0*dbk1 + ddbk1*x - ddbk2;
		break; 
	}
};
//==============================================================================//
// ������������ ������ ������ � ���������� ��������� �� �������� ������ �������
// GPU
//==============================================================================//
__device__ void kernalDE403( double t, int n_pl, int mode, double* x_pl, double ajd0, double delt0, double *d_EF )
{
	double jd50 = 2433282.5;
	double day_begin = -54770.0;
	double day_end = 59726.0;
	double span = 32.0;
	double daysec = 2.314814814815e-5;
	double *buffer;//[746];

	int c_pl;
	int j;
	int knot;
	int degree;
	double dif;
	double t_span;
	double arg;
	double day;

	double intp[11] = {8.0, 32.0, 16.0, 32.0, 32.0, 32.0, 32.0, 32.0, 32.0, 4.0, 32.0 };
	int p[11] = { 3, 147, 183, 273, 303, 330, 354, 378, 396, 414, 702 };
	int c[11] = { 12, 12, 15, 10, 9, 8, 8, 6, 6, 12, 15 };

	int first_rec = 1;
	// ����� � ��������� ���� � ������� TDB - ����������� �����
	day = ajd0-jd50 + (t + delt0)/86.4;
	//printf("input day = %f\n", day );
	// span - ��� ����� ����� �� ����������
	int rec_num = (int)((day-day_begin)/span);
	// ����� �����
	int clu = rec_num + first_rec;
	//printf( "clu = %d\n", clu );

	//char *DE_path = "data\\DE403.bin";
	//std::ifstream rfs( DE_path, std::ios::in | std::ios::binary);
	//for( int it = 0; it < clu; it++ )
	//	rfs.read( (char *)buffer, 746*sizeof( double ) );
	//rfs.close();

	//memcpy( buffer, &FileMemDE403[746*(clu-1)], 746*sizeof( double ) );
	// ��� GPU ��������� �� ���������� ������
	buffer = &d_EF[746*(clu-1)];

	//nint
	dif = (int)(2.0*(ajd0-jd50-buffer[1])/2.0)+(t+delt0)/86.40;
	// ����� ������������� ��� �������
	c_pl = c[n_pl];
	// ������� ��������
	degree = c_pl-1;
	// ������������ ��������
	t_span = intp[n_pl];
	// ����� ��������
	knot = (int)(dif/t_span);
	// ������� ��������� � ��������� ������������ [-1 1 ] 
	arg = 2.0*(dif/t_span-knot)-1.0;
	// ������ ������ ��������
	// 3* - ��� ����������
	j = p[n_pl] - 1 + 3*knot*c_pl;
	//printf("Pos index J = %d\n", j );

	t_span = daysec/t_span;
	switch( mode ) 
	{
	case 1:
		//do i = 1,3
		for( int i = 0; i < 3; i++ )
		{
			kernalcheb2(1, arg, degree, &buffer[j+i*c_pl], &x_pl[i], NULL, NULL );
			x_pl[i] = x_pl[i]*1.e-3;
		}
		break; 
	case 2:
		//do i = 1,3
		for( int i = 0; i < 3; i++ )
		{
			kernalcheb2(2, arg, degree, &buffer[j+i*c_pl], &x_pl[i], &x_pl[i+3], NULL);
			x_pl[i] = x_pl[i]*1.0e-3;
			x_pl[i+3] = x_pl[i+3]*t_span;
		}
		break; 
	case 3:
		//do i = 1,3
		for( int i = 0; i < 3; i++ )
		{
			kernalcheb2(3, arg, degree, &buffer[j+i*c_pl], &x_pl[i], &x_pl[i+3], &x_pl[i+6] );
			x_pl[i] = x_pl[i]*1.0e-3;
			x_pl[i+3] = x_pl[i+3]*t_span;
			x_pl[i+6] = x_pl[i+6]*t_span*t_span*1.0e+3;
		}
		break; 
	}
}
//==============================================================================//
// ��������� ���������� ������� ��������� ������.
// ���������� ������������ � ��������������� ��
// GPU
//==============================================================================//
__device__ void kernalplanets_update_geo( double t, double ajd0, double delt0, double *d_EF, double *PLCOORD )
{
	double EM_r[3];
	double M_r[3];

	kernalDE403( t, 2, 1, EM_r, ajd0, delt0, d_EF );	// �����+����
	kernalDE403( t, 9, 1, M_r, ajd0, delt0, d_EF );	// ���� (��������)

	//pln_coords(10,:) = M_r
	for( int it = 0; it < 3; it++ )				// ����
		PLCOORD[9*3+it]= M_r[it];

	//pln_coords(3,:) = EM_r-M_r/M_ME			// ����� 
	double M_ME = 82.300578;
	double POS_E[3];
	for( int it = 0; it < 3; it++ )
	{
		PLCOORD[2*3+it]= EM_r[it] - M_r[it]/M_ME;
		// ��������� ����� ������������ ������
		POS_E[it] = PLCOORD[2*3+it];
	}

	kernalDE403(t, 0, 1, &PLCOORD[0*3], ajd0, delt0, d_EF ); // ��������
	kernalDE403(t, 1, 1, &PLCOORD[1*3], ajd0, delt0, d_EF ); // ������
	kernalDE403(t, 3, 1, &PLCOORD[3*3], ajd0, delt0, d_EF ); // ����
	kernalDE403(t, 4, 1, &PLCOORD[4*3], ajd0, delt0, d_EF ); // ������
	kernalDE403(t, 5, 1, &PLCOORD[5*3], ajd0, delt0, d_EF ); // ������
	kernalDE403(t, 6, 1, &PLCOORD[6*3], ajd0, delt0, d_EF ); // ����
	kernalDE403(t, 7, 1, &PLCOORD[7*3], ajd0, delt0, d_EF ); // ������
	kernalDE403(t, 8, 1, &PLCOORD[8*3], ajd0, delt0, d_EF ); // ������

	// ��������� ������ ��� - ��������� �����
	for( int it = 0; it < 3; it++ )
		PLCOORD[10*3+it] = -PLCOORD[2*3+it];		// ������

	// ��������� ��� ������� ������� ��������� ������������ �����
	// 9 - ����
	// 10 - ������
	for( int i = 0; i < 9; i++ )
	{
		PLCOORD[i*3+0] = PLCOORD[i*3+0] - POS_E[0];
		PLCOORD[i*3+1] = PLCOORD[i*3+1] - POS_E[1];
		PLCOORD[i*3+2] = PLCOORD[i*3+2] - POS_E[2];
	}

	//t_planets_update = t;
}
//==============================================================================//
// ��������� ���������� ������� ��������� ������.
// ���������� ������������ � ����������������� ��
//==============================================================================//
//void InfluenceForce::planets_update_sol( double t, double ajd0, double delt0 )
//{
//	double EM_r[3];
//	double M_r[3];
//
//	kernalDE403( t, 2, 1, EM_r, ajd0, delt0);	// �����+����
//	kernalDE403( t, 9, 1, M_r, ajd0, delt0);	// ���� (��������)
//
//	//pln_coords(10,:) = M_r
//	for( int it = 0; it < 3; it++ )				// ����
//		PLCOORD[9*3+it]= M_r[it];
//
//	//pln_coords(3,:) = EM_r-M_r/M_ME ! ����� 
//	double M_ME = 82.300578;
//	for( int it = 0; it < 3; it++ )
//		PLCOORD[2*3+it]= EM_r[it] - M_r[it]/M_ME;
//
//	kernalDE403(t, 0, 1, &PLCOORD[0*3], ajd0, delt0 ); // ��������
//	kernalDE403(t, 1, 1, &PLCOORD[1*3], ajd0, delt0 ); // ������
//	kernalDE403(t, 3, 1, &PLCOORD[3*3], ajd0, delt0 ); // ����
//	kernalDE403(t, 4, 1, &PLCOORD[4*3], ajd0, delt0 ); // ������
//	kernalDE403(t, 5, 1, &PLCOORD[5*3], ajd0, delt0 ); // ������
//	kernalDE403(t, 6, 1, &PLCOORD[6*3], ajd0, delt0 ); // ����
//	kernalDE403(t, 7, 1, &PLCOORD[7*3], ajd0, delt0 ); // ������
//	kernalDE403(t, 8, 1, &PLCOORD[8*3], ajd0, delt0 ); // ������
//
//	//pln_coords(10,:) = pln_coords(10,:) + pln_coords(3,:)
//	for( int it = 0; it < 3; it++ )				// ����
//		PLCOORD[9*3+it] += PLCOORD[2*3+it];
//
//	for( int it = 0; it < 3; it++ )
//		PLCOORD[10*3+it] = 0;		// ������
//
//	//t_planets_update = t;
//}
//==============================================================================//
// ���������� �� ������
//==============================================================================//
__device__ void kernalplanets_grav( double *x, double *f_gr, double *PLCOORD  )
{
	double mu_plan[11] = {
	22.03208047245,		// ��������
	324.8587656142,		// ������
	398.6004415,		// �����
	42.828287,			// ����
	126712.59708,		// ������
	37939.51971,		// ������
	5780.158533417,		// ����
	6871.307771094,		// ������
	1.02086,			// ������
	4.90279914,			// ����
	132712439.935 };	// ������

	double BC[3];
	double BO[3];
	double BC_r;
	double BO_r;

	f_gr[0] = 0;
	f_gr[1] = 0;
	f_gr[2] = 0;

	for( int i = 0; i <= 10; i++ ) 
	{
		if( kernalpln_flag(i) )
		{
			// body - center
			kernalpln_coords( i, BC, PLCOORD );	

			//(BC,BC)
			BC_r = BC[0]*BC[0] + BC[1]*BC[1] + BC[2]*BC[2];	

			//BC_r.LT.1.d-12
			if ( BC_r < 1.0e-12 )	
			{
				//printf("BC_r < 1.0e-12 \n");
				// body - object
				BO[0] = BC[0]-x[0]; 
				BO[1] = BC[1]-x[1];
				BO[2] = BC[2]-x[2];
				// 1/(BO*BO)
				BO_r = 1.0/( BO[0]*BO[0] + BO[1]*BO[1] + BO[2]*BO[2] );
				BO_r = BO_r*sqrt(BO_r);

				f_gr[0] = f_gr[0] + mu_plan[i]*BO[0]*BO_r;
				f_gr[1] = f_gr[1] + mu_plan[i]*BO[1]*BO_r;
				f_gr[2] = f_gr[2] + mu_plan[i]*BO[2]*BO_r;
			}
			else
			{
				// body - object
				BO[0] = BC[0]-x[0]; 
				BO[1] = BC[1]-x[1];
				BO[2] = BC[2]-x[2];

				BC_r = 1.0/BC_r;
				BC_r = BC_r*sqrt(BC_r);

				// 1/(BO*BO)
				BO_r = 1.0/( BO[0]*BO[0] + BO[1]*BO[1] + BO[2]*BO[2] );
				BO_r = BO_r*sqrt(BO_r);

				f_gr[0] = f_gr[0] + mu_plan[i]*( BO[0]*BO_r - BC[0]*BC_r );
				f_gr[1] = f_gr[1] + mu_plan[i]*( BO[1]*BO_r - BC[1]*BC_r );
				f_gr[2] = f_gr[2] + mu_plan[i]*( BO[2]*BO_r - BC[2]*BC_r );
			}
		}
	}
};
//==============================================================================//