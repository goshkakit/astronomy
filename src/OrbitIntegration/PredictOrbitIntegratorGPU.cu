#include "hip/hip_runtime.h"
//==============================================================================//
// Andrianov N.G.
// opbit predict module
// Integration motion
// RK method
//==============================================================================//
#include "PredictOrbitSat.h" 

#ifdef GPUCOMPILE

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <math.h>
#include "cutilNP.h"



// name
namespace Orbit
{

		// ���������
static const double hh_tick[42] = {  
	500.0, 600.0, 640.0, 1500.0, 600.0, 640.0,  
	500.0, 660.0, 700.0, 1500.0, 700.0, 660.0,  
	500.0, 760.0, 760.0, 1500.0, 780.0, 740.0,  
	500.0, 800.0, 820.0, 1500.0, 800.0, 800.0,  
	500.0, 860.0, 860.0, 1500.0, 800.0, 860.0,  
	500.0, 900.0, 920.0, 1500.0, 900.0, 900.0,  
	500.0,1000.0, 980.0, 1500.0, 760.0, 900.0 };
static const double h_aa0[14] = { 
		.26862900E+02, .27459800E+02, .28639500E+02, .29641800E+02,  
		.30167100E+02, .29757800E+02, .30785400E+02,  
		.17878100E+02,-.25490900E+01,-.13959900E+02,-.23307900E+02,  
		-.14726400E+02,-.49120000E+01,-.54095200E+01 };
static const double h_aa1[14] = { 
		-.45167400E+00,-.46366800E+00,-.49098700E+00,-.51495700E+00,  
		-.52783700E+00,-.51791500E+00,-.54569500E+00,  
		-.13202500E+00, .14006400E-01, .84495100E-01, .13514100E+00,  
		.71325600E-01, .10832600E-01, .55074900E-02 };
static const double h_aa2[14] = { 
		.29039700E-02, .29740000E-02, .32064900E-02, .34192600E-02,  
		.35321100E-02, .34269900E-02, .37032800E-02,  
		.22771700E-03,-.16946000E-03,-.32887500E-03,-.42080200E-03,  
		-.22801500E-03,-.81054600E-04,-.37885100E-04 };
static const double h_aa3[14] = { 
		-.10695300E-04,-.10753000E-04,-.11681000E-04,-.12578500E-04,  
		-.13022700E-04,-.12413700E-04,-.13707200E-04,  
		-.22543000E-06, .32719600E-06, .50591800E-06, .57371700E-06,  
		.28487000E-06, .11571200E-06, .24808000E-07 };
static const double h_aa4[14] = { 
		.22159800E-07, .21705900E-07, .23684700E-07, .25727000E-07,  
		.26645500E-07, .24820900E-07, .28061400E-07,  
		.13357400E-09,-.28763000E-09,-.39229900E-09,-.40323800E-09,  
		-.17438300E-09,-.81329600E-10, .49218300E-11 };
static const double h_aa5[14] = { 
		-.24294100E-10,-.23024900E-10,-.25180900E-10,-.27587400E-10,  
		-.28543200E-10,-.25841300E-10,-.30018400E-10,  
		-.45045800E-13, .12262500E-12, .15227900E-12, .14284600E-12,  
		.50807100E-13, .30491300E-13,-.86501100E-14 };
static const double h_aa6[14] = { 
		.10992600E-13, .10012300E-13, .10953600E-13, .12109100E-13,  
		.12500900E-13, .10938300E-13, .13114200E-13,  
		.67208600E-17,-.20573600E-16,-.23557600E-16,-.20172600E-16,  
		-.53495500E-17,-.49498900E-17, .19849000E-17 };
static const double h_bb0[14] = { 
		.68789400E-01, .15073000E+00, .47945100E-01, .22344800E-01,  
		.32639100E-02,-.51474900E-01,-.10725500E+00,  
		.23158400E+02, .33273200E+02, .39196100E+02, .43246900E+02,  
		.49573800E+02, .11278000E+02,-.52618400E+02 };
static const double h_bb1[14] = { 
		-.28407700E-02,-.40088900E-02,-.23945300E-02,-.19798000E-02,  
		-.15986900E-02,-.92105900E-03,-.17434300E-03,  
		-.80214700E-01,-.11109900E+00,-.12352000E+00,-.12697300E+00,  
		-.13861300E+00, .14347800E-02, .21468900E+00 };
static const double h_bb2[14] = { 
		.18392200E-04, .24393700E-04, .17033500E-04, .15410100E-04,  
		.14044300E-04, .11514700E-04, .90275900E-05,  
		.10582400E-03, .14142100E-03, .14901500E-03, .14263700E-03,  
		.14785100E-03,-.36984600E-04,-.29488200E-03 };
static const double h_bb3[14] = { 
		.91960500E-08,-.99277200E-08,-.13162600E-08,-.23543000E-08,  
		-.30228700E-08,-.12290100E-08,-.31651200E-09,  
		-.61503600E-07,-.79495200E-07,-.79705000E-07,-.70998500E-07,  
		-.69636100E-07, .35831800E-07, .17117100E-06 };
static const double h_bb4[14] = { 
		-.41687300E-10,-.18223900E-10,-.17403200E-10,-.12499400E-10,  
		-.92016000E-11,-.81310400E-11,-.61400000E-11,  
		.13245300E-10, .16583600E-10, .15877200E-10, .13164600E-10,  
		.12159500E-10,-.99122500E-11,-.36058200E-10 };
static const double h_cc0[14] = { 
		-.10482500E+01,-.93106000E+00,-.82086700E+00,-.74404700E+00,  
		-.72247100E+00,-.68748200E+00,-.73998400E+00,  
		.50503400E+02, .61624000E+02, .53262300E+02, .18223600E+02,  
		-.31844200E+02,-.48720800E+02,-.14785900E+03 };
static const double h_cc1[14] = { 
		.16630500E-01, .14153700E-01, .11991600E-01, .10474300E-01,  
		.98031700E-02, .91659400E-02, .95285400E-02,  
		-.17054100E+00,-.19296700E+00,-.14434200E+00,-.84002400E-02,  
		.16832700E+00, .22299600E+00, .53165200E+00 };
static const double h_cc2[14] = { 
		-.92426300E-04,-.72986200E-04,-.57983500E-04,-.47854400E-04,  
		-.42524500E-04,-.38093200E-04,-.36272700E-04,  
		.21723200E-03, .22806100E-03, .14659000E-03,-.38800000E-04,  
		-.26260300E-03,-.32188400E-03,-.67193700E-03 };
static const double h_cc3[14] = { 
		.27238200E-06, .20029400E-06, .15070700E-06, .11851300E-06,  
		.99554400E-07, .85127500E-07, .73887000E-07,  
		-.12190200E-06,-.11871500E-06,-.64644300E-07, .43138400E-07,  
		.16545400E-06, .19149500E-06, .36478700E-06 };
static const double h_cc4[14] = { 
		-.24135500E-09,-.16200600E-09,-.11302600E-09,-.83149800E-10,  
		-.65517500E-10,-.52997200E-10,-.42390700E-10,  
		.25403700E-10, .22963800E-10, .10422700E-10,-.12383200E-10,  
		-.36935500E-10,-.40806700E-10,-.72626800E-10 };
static const double h_dd0[14] = { 
		-.35189900E+00,-.47813000E-01, .20981000E+00, .26517400E+00,  
		.23047000E+00, .17007400E+00, .88141000E-01,  
		-.35189900E+00,-.47813000E-01, .20981000E+00, .26517400E+00,  
		.23047000E+00, .17007400E+00, .88141000E-01 };
static const double h_dd1[14] = { 
		.57705600E-02, .38081300E-02, .26288100E-02, .27583600E-02,  
		.33833100E-02, .40613100E-02, .46825300E-02,  
		.57705600E-02, .38081300E-02, .26288100E-02, .27583600E-02,  
		.33833100E-02, .40613100E-02, .46825300E-02 };
static const double h_dd2[14] = { 
		.99581900E-06, .42277100E-05, .42437900E-05, .20866800E-05,  
		-.55230500E-06,-.28211400E-05,-.42460900E-05,  
		.99581900E-06, .42277100E-05, .42437900E-05, .20866800E-05,  
		-.55230500E-06,-.28211400E-05,-.42460900E-05 };
static const double h_dd3[14] = { 
		-.72532400E-08,-.86682600E-08,-.66732800E-08,-.36954300E-08,  
		-.82360700E-09, .13836900E-08, .25350900E-08,  
		-.72532400E-08,-.86682600E-08,-.66732800E-08,-.36954300E-08,  
		-.82360700E-09, .13836900E-08, .25350900E-08 };
static const double h_dd4[14] = { 
		.29759000E-11, .30671200E-11, .21349600E-11, .11186200E-11,  
		.22134900E-12,-.42790800E-12,-.72903100E-12,  
		.29759000E-11, .30671200E-11, .21349600E-11, .11186200E-11,  
		.22134900E-12,-.42790800E-12,-.72903100E-12 };
static const double h_ee0[14] = { 
		-.73159600E+00,-.75217500E+00,-.57047600E+00,-.94957300E+00,  
		-.96759800E+00,-.10227800E+01,-.75790300E+00,  
		.38619900E+02, .51249000E+02, .68474600E+02, .58422000E+02,  
		.72018800E+01, .21594800E+02,-.88407600E+02 };
static const double h_ee1[14] = { 
		.59734500E-02, .56592500E-02, .29580200E-02, .81312100E-02,  
		.84199100E-02, .92363300E-02, .60606800E-02,  
		-.13214700E+00,-.16737300E+00,-.21565900E+00,-.16666400E+00,  
		.21610900E-01,-.20223900E-01, .33851800E+00 };
static const double h_ee2[14] = { 
		-.58203700E-05, .18082000E-05, .16889600E-04,-.38781300E-05,  
		-.35850000E-05,-.61012800E-05, .78529600E-05,  
		.17541100E-03, .21183200E-03, .26227300E-03, .18548600E-03,  
		-.65288200E-04,-.17202900E-04,-.44558100E-03 };
static const double h_ee3[14] = { 
		.68463400E-07, .33382200E-07,-.47475600E-08, .23769400E-07,  
		.17480100E-07, .17821100E-07,-.97489100E-08,  
		-.10241700E-06,-.11822100E-06,-.14097200E-06,-.91234500E-07,  
		.53707700E-07, .28301700E-07, .25172900E-06 };
static const double h_ee4[14] = { 
		-.95048300E-10,-.51396500E-10,-.17271100E-10,-.27746900E-10,  
		-.19622100E-10,-.17007300E-10, .15837700E-11,  
		.22144600E-10, .24505500E-10, .28228500E-10, .16711800E-10,  
		-.14095000E-10,-.89448600E-11,-.52030000E-10 };
static const double h_ff1[7] = { 
		.54110000E+00, .55150000E+00, .55850000E+00, .55850000E+00,  
		.55850000E+00, .55850000E+00, .55850000E+00 };
static const double h_ee5[7] = { 
		-.20670000E+00,-.16971000E+00,-.14671000E+00,-.13150000E+00,  
		-.12091600E+00,-.11363000E+00,-.10444000E+00 };
static const double h_ee6[7] = { 
		.97533000E-01, .79830000E-01, .68808000E-01, .61603000E-01,  
		.56538000E-01, .53178000E-01, .48551000E-01 };
static const double h_ee7[7] = { 
		-.11817000E-01,-.94393000E-02,-.79836000E-02,-.70866000E-02,  
		-.64324000E-02,-.60436000E-02,-.53567000E-02 };
static const double h_ee8[7] = { 
		.16145000E-02, .12622000E-02, .10535000E-02, .92813000E-03,  
		.83723000E-03, .77982000E-03, .68809000E-03 };
static const double h_eet5[14] = { 
		-.20610000E+00,-.16927900E+00,-.14637700E+00,-.13121000E+00,  
		-.12067000E+00,-.11339900E+00,-.10424300E+00,  
		-.20610000E+00,-.16927900E+00,-.14637700E+00,-.13121000E+00,  
		-.12067000E+00,-.11339900E+00,-.10424300E+00 };
static const double h_eet6[14] = { 
		.94449000E-01, .77599000E-01, .67052000E-01, .60105000E-01,  
		.55232000E-01, .51994000E-01, .47573000E-01,  
		.94449000E-01, .77599000E-01, .67052000E-01, .60105000E-01,  
		.55232000E-01, .51994000E-01, .47573000E-01 };
static const double h_eet7[14] = { 
		-.87953000E-02,-.71375000E-02,-.60951000E-02,-.54388000E-02,  
		-.49580000E-02,-.46876000E-02,-.41711000E-02,  
		-.87953000E-02,-.71375000E-02,-.60951000E-02,-.54388000E-02,  
		-.49580000E-02,-.46876000E-02,-.41711000E-02 };
static const double h_eet8[14] = { 
		.88385000E-03, .69025000E-03, .57456000E-03, .50585000E-03,  
		.45512000E-03, .42548000E-03, .37068000E-03,  
		.88385000E-03, .69025000E-03, .57456000E-03, .50585000E-03,  
		.45512000E-03, .42548000E-03, .37068000E-03 };
static const double h_aal0[14] = { 
		-.40776800E+00,-.90273900E+00,-.73303700E+00,-.13144400E+01,  
		-.12002600E+01,-.15215800E+01,-.16766400E+01,  
		.48653600E+02, .54486700E+02, .60126700E+02, .47099600E+02,  
		.50617400E+02, .80194200E+01,-.15572800E+02 };
static const double h_aal1[14] = { 
		.14850600E-02, .82680300E-02, .52339600E-02, .13312400E-01,  
		.11408700E-01, .15704000E-01, .17719400E-01,  
		-.17029100E+00,-.17829800E+00,-.18314400E+00,-.12526000E+00,  
		-.12904700E+00, .18530200E-01, .93670400E-01 };
static const double h_aal2[14] = { 
		.12535700E-04,-.12544800E-04, .63566700E-05,-.25558500E-04,  
		-.14732400E-04,-.30285900E-04,-.36949800E-04,  
		.22624200E-03, .22272500E-03, .21248100E-03, .12635200E-03,  
		.12484200E-03,-.61473300E-04,-.14903600E-03 };
static const double h_aal3[14] = { 
		.37731100E-07, .61285300E-07, .10906500E-07, .54398100E-07,  
		.27804000E-07, .45766800E-07, .50913400E-07,  
		-.13203200E-06,-.12270000E-06,-.10849700E-06,-.55158400E-07,  
		-.52499300E-07, .49767400E-07, .94215100E-07 };
static const double h_aal4[14] = { 
		-.77895300E-10,-.70796600E-10,-.26142700E-10,-.43378400E-10,  
		-.22632000E-10,-.28292600E-10,-.28287800E-10,  
		.28519300E-10, .25131600E-10, .20571000E-10, .87527200E-11,  
		.80827200E-11,-.12616200E-10,-.20961000E-10 };

__device__ __constant__ int KKR[12];

__device__ __constant__ double Dca[120]; 
__device__ __constant__ double Dca1[40];
__device__ __constant__ double Dcw[76];
__device__ __constant__ double Dcg[6];
__device__ __constant__ double Dcc[16];

__device__ __constant__ double CUtick[42];

__device__ __constant__ double CUaal0[14];
__device__ __constant__ double CUaal1[14];
__device__ __constant__ double CUaal2[14];
__device__ __constant__ double CUaal3[14];
__device__ __constant__ double CUaal4[14];

__device__ __constant__ double CUaa0[14];
__device__ __constant__ double CUaa1[14];
__device__ __constant__ double CUaa2[14];
__device__ __constant__ double CUaa3[14];
__device__ __constant__ double CUaa4[14];
__device__ __constant__ double CUaa5[14];
__device__ __constant__ double CUaa6[14];

__device__ __constant__ double CUbb0[14];
__device__ __constant__ double CUbb1[14];
__device__ __constant__ double CUbb2[14];
__device__ __constant__ double CUbb3[14];
__device__ __constant__ double CUbb4[14];

__device__ __constant__ double CUcc0[14];
__device__ __constant__ double CUcc1[14];
__device__ __constant__ double CUcc2[14];
__device__ __constant__ double CUcc3[14];
__device__ __constant__ double CUcc4[14];

__device__ __constant__ double CUdd0[14];
__device__ __constant__ double CUdd1[14];
__device__ __constant__ double CUdd2[14];
__device__ __constant__ double CUdd3[14];
__device__ __constant__ double CUdd4[14];

__device__ __constant__ double CUee0[14];
__device__ __constant__ double CUee1[14];
__device__ __constant__ double CUee2[14];
__device__ __constant__ double CUee3[14];
__device__ __constant__ double CUee4[14];
		
__device__ __constant__ double CUee5[7];
__device__ __constant__ double CUee6[7];
__device__ __constant__ double CUee7[7];
__device__ __constant__ double CUee8[7];
__device__ __constant__ double CUff1[7];

__device__ __constant__ double CUeet5[14];
__device__ __constant__ double CUeet6[14];
__device__ __constant__ double CUeet7[14];
__device__ __constant__ double CUeet8[14];

// ���������� ����������� �� �������
#include "cuInfluencePlanet.cu"
#include "cuInfluenceEarthRotation.cu"
#include "cuInfluenceEGM96.cu"
#include "cuInfluenceSun.cu"
#include "cuInfluenceAtmosphere.cu"

//=====================================================================//
// ������� � ��������
//=====================================================================//
struct OrbitArrayPointsGPU
{
	// ������ �������
	unsigned int mem_size;
	// ����� �����
	int N_hlist;
	// ����� �����
	int Length; 

	// ���������
	double **h_array_list; 
	double **d_array_list;

	void InitArrayList( int N, int L )
	{
		// ������ ������ ��������
		mem_size = sizeof(double)*(L*4+1);

		// ��������� ������ �� ����������
		N_hlist = N;
		Length = L;

		h_array_list = new double*[N_hlist];
		cutilSafeCall( hipMalloc((void**)&d_array_list, N_hlist*sizeof( double* ) ));

		// ��������� ������� ������
		for ( int i = 0; i < N_hlist; i++ )
		{
			// allocate memory for one arrays on the device
			cutilSafeCall( hipMalloc( (void**)&h_array_list[i], mem_size ) );
		}
		// ����������� �� ���������� ������� � ����������� �� ������
		cutilSafeCall( hipMemcpy( d_array_list, h_array_list, N_hlist*sizeof(double*), hipMemcpyHostToDevice ));
	};

	void FreeArrayList()
	{
		// ������� ������� � �������
		for (int i = 0; i < N_hlist; i++)
		{
			hipFree( h_array_list[i] );
		}
		hipFree( d_array_list );

		delete h_array_list;
	};

	void CopyFromGPU( Orbit::OrbitArrayPointsCPU &outArr )
	{
		for (int i = 0; i < N_hlist; i++)
		{
			cutilSafeCall( hipMemcpy( outArr.array_list[i], h_array_list[i], mem_size, hipMemcpyDeviceToHost ) );
		}
	};
};
//=====================================================================//
// ��������� ������ ��� GPU
//=====================================================================//
struct gpuOrbitPoint
{
	int NY;
	int Block;
	int sizeMem;
	int sizeMemOne;
	int sizeMemT;

	double *TP0;
	double *X0;
	double *FP0;
	double *FP1;
	double *FP2;
	double *FP3;
	double *FP4;
	double *FP5;
	double *FP6;
	double *FP7;

	double ajd0;
	double delt0;
	double Satm;
	double Ssun;

	CurrentIntegrateParam *Iparam;

	// ��������� �� ���������
	double *d_EF;
	// ��������� �� ������� �������� �������
	double *dT_finals_tab;
	int *dT_finals_n;
	// ������� �������� ��� �������
	double *dNUT_AMPL;
	double *dNUT_ARG;
	// ��� ��������
	double *d_Garmonic;

	void AllocMemory( int ny, int block )
	{
		NY = ny;
		Block = block;
		sizeMem = block*NY*sizeof(double);
		sizeMemOne = NY*sizeof(double);
		sizeMemT = block*sizeof(double);

		// ������ ��� ���� ��������
		hipMalloc((void**)&TP0, sizeMemT );
	
		hipMalloc((void**)&X0, sizeMem );

		hipMalloc((void**)&FP0, sizeMem );
		hipMalloc((void**)&FP1, sizeMem );
		hipMalloc((void**)&FP2, sizeMem );
		hipMalloc((void**)&FP3, sizeMem );
		hipMalloc((void**)&FP4, sizeMem );
		hipMalloc((void**)&FP5, sizeMem );
		hipMalloc((void**)&FP6, sizeMem );
		hipMalloc((void**)&FP7, sizeMem );

		hipMalloc((void**)&Iparam, block*sizeof( CurrentIntegrateParam ) );
	};

	void DeleteMemory()
	{
		hipFree(TP0);
		hipFree(X0);
		hipFree(FP0);
		hipFree(FP1);
		hipFree(FP2);
		hipFree(FP3);
		hipFree(FP4);
		hipFree(FP5);
		hipFree(FP6);
		hipFree(FP7);
		hipFree(Iparam);
	};

	void CopyToGPU( Orbit::SatelliteArray &inListSat )
	{
		cutilSafeCall( hipMemcpy( TP0, inListSat.TP0, sizeMemT, hipMemcpyHostToDevice ) );
		// ������� �������� ��������� �������� - ���������, ��������
		// �������� ���������������� �1 �2..... y1 y2.....
		cutilSafeCall( hipMemcpy( X0, inListSat.X0, sizeMem, hipMemcpyHostToDevice ) );
		cutilSafeCall( hipMemcpy( FP0, inListSat.X0, sizeMem, hipMemcpyHostToDevice ) );

		//hipMemcpy( FP1, inOP.FP1, sizeMem, hipMemcpyHostToDevice );
		//hipMemcpy( FP2, inOP.FP2, sizeMem, hipMemcpyHostToDevice );
		//hipMemcpy( FP3, inOP.FP3, sizeMem, hipMemcpyHostToDevice );
		//hipMemcpy( FP4, inOP.FP4, sizeMem, hipMemcpyHostToDevice );
		//hipMemcpy( FP5, inOP.FP5, sizeMem, hipMemcpyHostToDevice );
		//hipMemcpy( FP6, inOP.FP6, sizeMem, hipMemcpyHostToDevice );
		//hipMemcpy( FP7, inOP.FP7, sizeMem, hipMemcpyHostToDevice );
	};

	void CopyFromGPU( Orbit::SatelliteArray &outListSat )
	{
		// �������� ���������������� �1 �2..... y1 y2.....
		cutilSafeCall( hipMemcpy( outListSat.X0, X0, sizeMem, hipMemcpyDeviceToHost ) );

		//hipMemcpy( outOP.FP0, FP0, sizeMem, hipMemcpyDeviceToHost );
		//hipMemcpy( outOP.FP1, FP1, sizeMem, hipMemcpyDeviceToHost );
		//hipMemcpy( outOP.FP2, FP2, sizeMem, hipMemcpyDeviceToHost );
		//hipMemcpy( outOP.FP3, FP3, sizeMem, hipMemcpyDeviceToHost );
		//hipMemcpy( outOP.FP4, FP4, sizeMem, hipMemcpyDeviceToHost );
		//hipMemcpy( outOP.FP5, FP5, sizeMem, hipMemcpyDeviceToHost );
		//hipMemcpy( outOP.FP6, FP6, sizeMem, hipMemcpyDeviceToHost );
		//hipMemcpy( outOP.FP7, FP7, sizeMem, hipMemcpyDeviceToHost);
	};
};

//=====================================================================//
// ������� ������ ����� �� gpu
//=====================================================================//
// ������� ���������� � ���������� ������ � ������ �� ����������
//__device__ void kernalFFxyzW(double t, double *xt, double *fx )
//{
//	double mm = 9.822*6370000.0*6370000.0;
//	int idt = blockDim.x*blockIdx.x + threadIdx.x;
//
//	double x = xt[idt + 0*CU_BlockXYZ];
//	double y = xt[idt + 1*CU_BlockXYZ];
//	double z = xt[idt + 2*CU_BlockXYZ];
//
//	double r = sqrt( x*x + y*y + z*z );
//	double IR3 = 1.0/(r*r*r);
//
//	fx[idt + 0*CU_BlockXYZ] = xt[idt + 3*CU_BlockXYZ];
//	fx[idt + 1*CU_BlockXYZ] = xt[idt + 4*CU_BlockXYZ];
//	fx[idt + 2*CU_BlockXYZ] = xt[idt + 5*CU_BlockXYZ];
//
//	fx[idt + 3*CU_BlockXYZ] = -x*mm*IR3;
//	fx[idt + 4*CU_BlockXYZ] = -y*mm*IR3;
//	fx[idt + 5*CU_BlockXYZ] = -z*mm*IR3;
//};

// ������� ���������� � ���������� ������
//__device__ void kernalFFxyzRK(double t, double *xt, double *fx)
//{
//	int idt = blockDim.x*blockIdx.x + threadIdx.x;
//
//	double mm = 9.822*6370000.0*6370000.0;
//
//	double x = xt[0];
//	double y = xt[1];
//	double z = xt[2];
//
//	double vx = xt[3];
//	double vy = xt[4];
//	double vz = xt[5];
//
//	double r = sqrt( x*x + y*y + z*z );
//	double R3 = r*r*r;
//
//	double ffx = vx;
//	double ffy = vy;
//	double ffz = vz;
//	double ffvx = -x*mm/R3;
//	double ffvy = -y*mm/R3;
//	double ffvz = -z*mm/R3;
//
//	fx[idt + 0*CU_BlockXYZ] = ffx;
//	fx[idt + 1*CU_BlockXYZ] = ffy;
//	fx[idt + 2*CU_BlockXYZ] = ffz;
//
//	fx[idt + 3*CU_BlockXYZ] = ffvx;
//	fx[idt + 4*CU_BlockXYZ] = ffvy;
//	fx[idt + 5*CU_BlockXYZ] = ffvz;
//};
//// ������� �������
//__device__ void kernalFFxyz(double t, double *xt, double *fx )
//{
//	double mm = 9.822*6370000.0*6370000.0;
//
//	double x = xt[0];
//	double y = xt[1];
//	double z = xt[2];
//
//	double r = sqrt( x*x + y*y + z*z );
//	double IR3 = 1.0/(r*r*r);
//
//	fx[0] = xt[3];
//	fx[1] = xt[4];
//	fx[2] = xt[5];
//
//	fx[3] = -x*mm*IR3;
//	fx[4] = -y*mm*IR3;
//	fx[5] = -z*mm*IR3;
//};
//=====================================================================//
// ���������� ����������
// ������� ���������� � ���������� ������
//=====================================================================//
__device__ void  kernalFFxyzRK( double t, double *x, double *f, gpuOrbitPoint *OP )
{
	int idt = blockDim.x*blockIdx.x + threadIdx.x;

	double PLCOORD[11*3];

	double f_gr[3];
	double A_rot[9];

	double x_g[6];
	double f_hrm[3];
	double f_ah[3];
	

	//double S_ajd0 = 2456192.50;
	//double S_delt0 = -10.73281600;
	double S_ajd0 = OP->ajd0;
	double S_delt0 = OP->delt0;
	//double sp_q = 0.5E-05;
	//double sigma_up = 0.3E-2;
	double sp_q = OP->Ssun;
	double sigma_up =  OP->Satm;

	//A_rot[0] = 1;
	//A_rot[1] = 0;
	//A_rot[2] = 0;

	//A_rot[3] = 0;
	//A_rot[4] = 1;
	//A_rot[5] = 0;

	//A_rot[6] = 0;
	//A_rot[7] = 0;
	//A_rot[8] = 1;

	// ������� ��������
	kernaliers_mat( t, A_rot, S_ajd0, S_delt0, OP->dNUT_ARG, OP->dNUT_AMPL, OP->dT_finals_tab, OP->dT_finals_n[0] );
	//if( idt == SetOrig-1 )
	//{
	//	for( int j = 0; j < 9; j++ )
	//		printf( "%.10f ", A_rot[j] );
	//	printf( "%.10e %.10e %.10e\n", t, S_ajd0, S_delt0);
	//}
	// ��������� ������
	kernalplanets_update_geo( t, S_ajd0, S_delt0, OP->d_EF, PLCOORD );

	f[idt + 0*CU_BlockXYZ] = x[3];
	f[idt + 1*CU_BlockXYZ] = x[4];
	f[idt + 2*CU_BlockXYZ] = x[5];

	// ���������� ������
	kernalplanets_grav( x, f_gr, PLCOORD );

	// ������� ������� � ������� ��������� �����
	kernalstate_to_itrf( t, x, x_g, A_rot );

	// ������� ��������
	kernalGetF_Harm_egm96( x_g, 75, f_hrm, OP->d_Garmonic );
	
	// ��������� ��������
	double f_sp[3];
	double pln_coords11[3];
	kernalpln_coords( 10, pln_coords11, PLCOORD );
	kernalsp_cannonballForce( x, sp_q, f_sp, pln_coords11 );

	//--- ������� ��������� ---//
	double f_atm[3];
	kernalAtm_drag( x_g, t, f_atm, sigma_up, S_ajd0, S_delt0 );
	f_hrm[0] = f_atm[0] + f_hrm[0];
	f_hrm[1] = f_atm[1] + f_hrm[1];
	f_hrm[2] = f_atm[2] + f_hrm[2];
	//------------------------//

	// ������� ������� ����������� �������� � ���������
	double invA[9];
	kernaltranspose( A_rot, invA ); 
	kernalmatVecMul( invA, f_hrm, f_ah );
	
	//if( idt == SetOrig-1 )
	//{
	//	printf( "%.10e %.10e %.10e\n", f_ah[0], f_ah[1], f_ah[2] );
	//}
	// ������������ �����������
	f[idt + 3*CU_BlockXYZ] = f_gr[0] + f_ah[0] + f_sp[0];
	f[idt + 4*CU_BlockXYZ] = f_gr[1] + f_ah[1] + f_sp[1];
	f[idt + 5*CU_BlockXYZ] = f_gr[2] + f_ah[2] + f_sp[2];
}
//=====================================================================//
// ���������� ����������
//=====================================================================//
__device__ void  kernalFFxyz( double t, double *x, double *f, gpuOrbitPoint *OP )
{
	int idt = blockDim.x*blockIdx.x + threadIdx.x;

	double PLCOORD[11*3];
	double f_gr[3];
	double A_rot[9];

	double x_g[6];
	double f_hrm[3];
	double f_ah[3];

	//double S_ajd0 = 2456192.50;
	//double S_delt0 = -10.73281600;
	double S_ajd0 = OP->ajd0;
	double S_delt0 = OP->delt0;
	//double sp_q = 0.5E-05;
	//double sigma_up = 0.3E-2;
	double sp_q = OP->Ssun;
	double sigma_up =  OP->Satm;

	//A_rot[0] = 1;
	//A_rot[1] = 0;
	//A_rot[2] = 0;

	//A_rot[3] = 0;
	//A_rot[4] = 1;
	//A_rot[5] = 0;

	//A_rot[6] = 0;
	//A_rot[7] = 0;
	//A_rot[8] = 1;
	// ������� ��������
	kernaliers_mat( t, A_rot, S_ajd0, S_delt0, OP->dNUT_ARG, OP->dNUT_AMPL, OP->dT_finals_tab, OP->dT_finals_n[0] );

	// ��������� ������
	kernalplanets_update_geo( t, S_ajd0, S_delt0, OP->d_EF, PLCOORD );

	f[0] = x[3];
	f[1] = x[4];
	f[2] = x[5];

	// ���������� ������
	kernalplanets_grav( x, f_gr, PLCOORD );

	// ������� ������� � ������� ��������� �����
	kernalstate_to_itrf( t, x, x_g, A_rot );

	// ������� ��������
	kernalGetF_Harm_egm96( x_g, 75, f_hrm, OP->d_Garmonic );
	
	// ��������� ��������
	double f_sp[3];
	double pln_coords11[3];
	kernalpln_coords( 10, pln_coords11, PLCOORD );
	kernalsp_cannonballForce( x, sp_q, f_sp, pln_coords11 );

	//--- ������� ��������� ---//
	double f_atm[3];
	kernalAtm_drag( x_g, t, f_atm, sigma_up, S_ajd0, S_delt0 );
	f_hrm[0] = f_atm[0] + f_hrm[0];
	f_hrm[1] = f_atm[1] + f_hrm[1];
	f_hrm[2] = f_atm[2] + f_hrm[2];
	//------------------------//

	// ������� ������� ����������� �������� � ���������
	double invA[9];
	kernaltranspose( A_rot, invA ); 
	kernalmatVecMul( invA, f_hrm, f_ah );

	// ������������ �����������
	f[3] = f_gr[0] + f_ah[0] + f_sp[0];
	f[4] = f_gr[1] + f_ah[1] + f_sp[1];
	f[5] = f_gr[2] + f_ah[2] + f_sp[2];
}

//=====================================================================//
// ���������� ��������� ����� ������� �����-�����
//=====================================================================//
__global__ void kernalGetStartPoint( gpuOrbitPoint *OP, double h )
{
	int idt = blockDim.x*blockIdx.x + threadIdx.x;

	//int kkr[12] = { 3,4,5,6,1,2,4,3,2,1,6,5 };

	double g[6] = {	0.069431844202973712388026755553595247452137,
					0.330009478207571867598667120448377656399712,
					0.669990521792428132401332879551622343600287,
					0.930568155797026287611973244446404752547862,
					1.069431844202973712388026755553595247452137,
					1.330009478207571867598667120448377656399712 };
	int NY;
	double X0[6];
	double TP0;
	// �������� ��������
	// x1 x2 x3 ..... y1 y2 y3 ..... z1 z2 z3 .... vx . vy . vz
	// ����� �������� �� block
	X0[0] = OP->X0[idt + 0*CU_BlockXYZ];
	X0[1] = OP->X0[idt + 1*CU_BlockXYZ];
	X0[2] = OP->X0[idt + 2*CU_BlockXYZ];
	X0[3] = OP->X0[idt + 3*CU_BlockXYZ];
	X0[4] = OP->X0[idt + 4*CU_BlockXYZ];
	X0[5] = OP->X0[idt + 5*CU_BlockXYZ];
	TP0 = OP->TP0[idt];

	NY = OP->NY;
	
	//===============================================================//
	//if( kp == 1 || kp == 2 )
	// MAIN cycle of starting procedure
	// TR,HR - current time & current step
	// P  = F1+2*F2+2*F3+F4
	// fun(double t, double *xt, double *fx )
	double TPN;
	double XN[6];	// ������ ���������
	double FN1[6];	// �������� �������
	double FN2[6];	// �������� �������

	for (int j = 1; j <= 6; ++j)
	{
		//int kk = kkr[j - 1];				// nodes following: 3,4,5,6,1,2 
		int kk = KKR[j - 1];
		double hr = h * (1.0 - g[kk - 1]);	// step
		//kk = kkr[j + 5];
		kk = KKR[j + 5];
		double tr = TP0 + hr;				// next time step

		kernalFFxyz( TP0, X0, FN1, OP );
		//^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^

		// F2=F(T0+1/2H,X0+1/2HF1) 
		for( int it = 0; it < NY; it++ )
			XN[it] = hr * FN1[it]*0.5 + X0[it];
		TPN = TP0 + hr*0.5;
		kernalFFxyz( TPN, XN, FN2, OP );
		//^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^

		// F3=F(T0+1/2H,X0+1/2HF2)
		for( int it = 0; it < NY; it++ )
		{
			XN[it] = hr * FN2[it]*0.5 + X0[it];
			FN1[it] += FN2[it]*2.0;
		}
		TPN = TP0 + hr*0.5;
		kernalFFxyz( TPN, XN, FN2, OP );
		//^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^

		// F4=F(T0+H,X0+HF3)
		for( int it = 0; it < NY; it++ )
		{
			XN[it] = hr * FN2[it] + X0[it];
			FN1[it] += FN2[it]*2.0;
		}
		TPN = TP0 + hr;
		kernalFFxyz( TPN, XN, FN2, OP );
		//^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^^

		// X1=X0+1/6H(F1+2F2+2F3+F4)
		for( int it = 0; it < NY; it++ )
			XN[it] = X0[it] +  hr / 6.0 *( FN1[it] + FN2[it] );

		// ���������� �����
		// 4,3,2,1,6,5 []
		if( j == 1)	kernalFFxyzRK(tr, XN, OP->FP4, OP );
		if( j == 2)	kernalFFxyzRK(tr, XN, OP->FP3, OP );
		if( j == 3)	kernalFFxyzRK(tr, XN, OP->FP2, OP );
		if( j == 4)	kernalFFxyzRK(tr, XN, OP->FP1, OP );
		if( j == 5)	kernalFFxyzRK(tr, XN, OP->FP6, OP );
		if( j == 6)	kernalFFxyzRK(tr, XN, OP->FP5, OP );
	}
	//=================================================================//
}
//=====================================================================//
// ��������� ���������
//=====================================================================//
__global__ void kernalOrbitInitParam( gpuOrbitPoint *OP, double initStep )
{
	int idt = blockDim.x*blockIdx.x + threadIdx.x;
	
	OP->Iparam[idt].hh = initStep;
	OP->Iparam[idt].NY = 6;	
	OP->Iparam[idt].NP = 0;
	OP->Iparam[idt].IP9 = 2;

	OP->Iparam[idt].l = 0;
	OP->Iparam[idt].l1 = 0;
	OP->Iparam[idt].l2 = 0;
	OP->Iparam[idt].ltek = 0;

	OP->Iparam[idt].kkbeg = 1;
	OP->Iparam[idt].InvertStart = false;
}
//=====================================================================//
// ������� ������
//=====================================================================//
__global__ void kernalOrbitPredict( gpuOrbitPoint *OP, double h, double t, double e, double **d_array, bool SavePoint ) //double *indc_a, double *indc_a1, double *indc_w, double *indc_g, double *indc_c
{
	int tx = threadIdx.x;
	int idt = blockDim.x*blockIdx.x + threadIdx.x;

	__shared__ int la[5];		// = { 0,24,48,72,96 };
	__shared__ int la1[5];		// = { 0,8,16,24,32 };
	__shared__ int la2[5];		// = { 0,24,24,24,48 };
	__shared__ double alim[2];	// = { 0.0403536069, 1.0 };
	__shared__ int isinv[5];	// = { 5,2,2,2,1 };
	__shared__ int isge[5];		// = { 4,5,5,5,3 };
	__shared__ int isle[5];		// = { 4,1,1,1,3 };
	__shared__ int iseq[5];		// = { 4,2,2,2,3 };

	// ptr to orbit for this thread
	
	
	//__shared__ double dc_a[120];
	//__shared__ double dc_a1[40];
	//__shared__ double dc_w[76];
	//__shared__ double dc_g[6];
	//__shared__ double dc_c[16];

	//__shared__ double *X0;
	//__shared__ double *FP0;
	//__shared__ double *FP1;
	//__shared__ double *FP2;
	//__shared__ double *FP3;
	//__shared__ double *FP4;
	//__shared__ double *FP5;
	//__shared__ double *FP6;
	//__shared__ double *FP7;

	double* s_array;
	if( SavePoint )
		s_array = d_array[idt];

	// ������������� ��������
	if( tx == 0 )
	{
		la[0] = 0;
		la[1] = 24;
		la[2] = 48;
		la[3] = 72;
		la[4] = 96;

		la1[0] = 0;
		la1[1] = 8;
		la1[2] = 16;
		la1[3] = 24;
		la1[4] = 32;
	
		la2[0] = 0;
		la2[1] = 24;
		la2[2] = 24;
		la2[3] = 24;
		la2[4] = 48;

		alim[0] = 0.0403536069;
		alim[1] = 1.0;

		isinv[0] = 5;
		isinv[1] = 2;
		isinv[2] = 2;
		isinv[3] = 2;
		isinv[4] = 1;

		isge[0] = 4;
		isge[1] = 5;
		isge[2] = 5;
		isge[3] = 5;
		isge[4] = 3;

		isle[0] = 4;
		isle[1] = 1;
		isle[2] = 1;
		isle[3] = 1;
		isle[4] = 3;

		iseq[0] = 4;
		iseq[1] = 2;
		iseq[2] = 2;
		iseq[3] = 2;
		iseq[4] = 3;

		//X0 = OP->X0;
		//FP0 = OP->FP0;
		//FP1 = OP->FP1;
		//FP2 = OP->FP2;
		//FP3 = OP->FP3;
		//FP4 = OP->FP4;
		//FP5 = OP->FP5;
		//FP6 = OP->FP6;
		//FP7 = OP->FP7;
	}

	// �������� �������������
	//if( tx < 120 )
	//	dc_a[tx] = indc_a[tx];
	//if( tx < 40 )
	//	dc_a1[tx] = indc_a1[tx];
	//if( tx < 76 )
	//	dc_w[tx] = indc_w[tx];
	//if( tx < 6 )
	//	dc_g[tx] = indc_g[tx];
	//if( tx < 16 )
	//	dc_c[tx] = indc_c[tx];

	__syncthreads ();

	//// ��������� ����������
	//// �� ������� �����
	double X0[6];
	double FP0[6];
	double FP1[6];
	double FP2[6];
	double FP3[6];
	double FP4[6];
	double FP5[6];
	double FP6[6];
	double FP7[6];
	
	//// ��������� ������ ��� ������� ������
	//// � ����� ����� �������� ���, ��� ��� ���� �������� � ���������� �������
	//// ����� ����������
	for( int it = 0; it < 6; it++ )
	{
		X0[it] = OP->X0[idt + it*CU_BlockXYZ];
		FP0[it] = OP->FP0[idt + it*CU_BlockXYZ];
		FP1[it] = OP->FP1[idt + it*CU_BlockXYZ];
		FP2[it] = OP->FP2[idt + it*CU_BlockXYZ];
		FP3[it] = OP->FP3[idt + it*CU_BlockXYZ];
		FP4[it] = OP->FP4[idt + it*CU_BlockXYZ];
		FP5[it] = OP->FP5[idt + it*CU_BlockXYZ];
		FP6[it] = OP->FP6[idt + it*CU_BlockXYZ];
		FP7[it] = OP->FP7[idt + it*CU_BlockXYZ];
	}

	double TP0 = OP->TP0[idt];
	double TPN = 0;
	//===============================================================//
	// ��������� ��� ����������
	//double hh = h;
	//int NY = 6;	
	//int NP = 0;
	//int IP9 = 2;

	//int l = 0;
	//int l1 = 0;
	//int l2 = 0;
	//int ltek = 0;

	//int kkbeg = 1;
	//double delt;
	//double rr1;
	//double rr2;
	//double rotn;
	//double dd1;
	//bool InvertStart = false;

	double hh = OP->Iparam[idt].hh;
	int NY = OP->Iparam[idt].NY;	
	int NP = OP->Iparam[idt].NP;
	int IP9 = OP->Iparam[idt].IP9;

	int l = OP->Iparam[idt].l;
	int l1 = OP->Iparam[idt].l1;
	int l2 = OP->Iparam[idt].l2;
	int ltek = OP->Iparam[idt].ltek;

	int kkbeg = OP->Iparam[idt].kkbeg;
	double delt = OP->Iparam[idt].delt;
	double rr1 = OP->Iparam[idt].rr1;
	double rr2 = OP->Iparam[idt].rr2;
	double rotn = OP->Iparam[idt].rotn;
	double dd1 = OP->Iparam[idt].dd1;
	bool InvertStart = OP->Iparam[idt].InvertStart;
	//===============================================================//

	//!!!!! ����� ��������
	idt = 0;
	int BlockXYZi = 1;
	//===============================================================//

	//------------------------------------------------//
	if( SavePoint )
	{
		// ������ ���������� ���������
		int itwrite = 1;
		double w_t = TP0;
		double w_x = X0[0];
		double w_y = X0[1];
		double w_z = X0[2];
		s_array[ itwrite ] = w_t;
		s_array[ itwrite + 1] = w_x;
		s_array[ itwrite + 2] = w_y;
		s_array[ itwrite + 3] = w_z;
	}
	//------------------------------------------------//

	if (hh * (t - TP0) < 0.0)
	{ 
		//printf("direct change\n");
		// direct change.

		for( int it = 0; it < NY; it++ )
		{
			int itt = idt + it*BlockXYZi;

			double tmp = FP1[itt];
			FP1[itt] = FP6[itt];
			FP6[itt] = tmp;

			tmp = FP2[itt];
			FP2[itt] = FP5[itt];
			FP5[itt] = tmp;

			tmp = FP3[itt];
			FP3[itt] = FP4[itt];
			FP4[itt] = tmp;
		}

		// step change
		hh = -hh;
		if (IP9 == 1) {	hh /= 0.7;	}
		if (IP9 == 5) {	hh *= 0.7;	}

		// case nuber cange 
		IP9 = isinv[IP9 - 1];

		//  2 steps of extrapolation
		kkbeg = 3;

		//goto L1400;
		InvertStart = true;
	}
	//===============================================================//

	int isstep = 0;
	// main cyrcle
	while( 1 )
	{
		if( InvertStart == false )
		{
			//===============================================================//
			//  S4: time overflow 
			if (hh * (t - TP0 - hh) <= 0.0 )
				break; 
			
			//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
			if( isstep > 2990 )
				break;

			// if overflow absent => step of implicit RK-method 
			// A5: step of implicit RK-method + error estimation
			TP0 +=hh;

			// DELT = maximum error
			// RR1,RR2 - working values
			// ROTH - error for i-th equation
			delt = 0.0;
			for (int in = 0; in < NY; in++ )
			{
				int itt = idt + in*BlockXYZi;
				//rr2 = FP3[itt]*dc_w[72] + FP4[itt]*dc_w[73] + FP5[itt]*dc_w[74] + FP6[itt]*dc_w[75];
				rr2 = FP3[itt]*Dcw[72] + FP4[itt]*Dcw[73] + FP5[itt]*Dcw[74] + FP6[itt]*Dcw[75];

				//step counter starting setting NP = 0;
				if (NP == 0 || kkbeg == 3 ){	
					FP0[itt] +=  hh * rr2;
					continue;
				}

				//rr1 = FP7[itt] + FP2[itt]*dc_a1[l1 + 4] + FP3[itt]*dc_a1[l1 + 5] + FP4[itt]*dc_a1[l1 + 6] + FP5[itt]*dc_a1[l1 + 7];
				rr1 = FP7[itt] + FP2[itt]*Dca1[l1 + 4] + FP3[itt]*Dca1[l1 + 5] + FP4[itt]*Dca1[l1 + 6] + FP5[itt]*Dca1[l1 + 7];
				dd1 = hh * (rr1 - rr2) / e;
				rotn = fabs(dd1);
				if (rotn >= delt){	delt = rotn;}

				FP0[itt] +=  hh * rr2;
			}
			NP++;
			//===============================================================//

			//===============================================================//
			//STEP: increase, decrease or do not change? 
			if ( NP == 1 || kkbeg == 3 ){ 
				IP9 = iseq[IP9 - 1];
				kkbeg = 1;
			}
			else if(alim[0] >= delt){	
				// A6:step increase 
				if (IP9 == 1) {	hh = hh; }
				if (IP9 == 2) {	hh /= 0.7;	}
				if (IP9 == 3) {	hh /= 0.7;	}
				if (IP9 == 4) {	hh /= 0.7;	}
				if (IP9 == 5) {	hh = hh;	}
				IP9 = isge[IP9 - 1];
			}
			else if (alim[1] <= delt) {	
				// A7:step decrease 
				if (IP9 == 1) {	hh = hh; }
				if (IP9 == 2) {	hh *= 0.7;	}
				if (IP9 == 3) {	hh *= 0.7;	}
				if (IP9 == 4) {	hh *= 0.7;	}
				if (IP9 == 5) {	hh = hh;	}
				IP9 = isle[IP9 - 1];
			}
			else
			{
				IP9 = iseq[IP9 - 1];
				kkbeg = 1;
			}
			//===============================================================//
		}

		InvertStart = false;
		//===============================================================//
		// A4: extrapolation 2|4 points + interpolation 
		l = la[IP9 - 1];
		l1 = la1[IP9 - 1];
		l2 = la2[IP9 - 1];
		//===============================================================//

		//===============================================================//
		for(int ii = 0; ii < NY; ii++ ) 
			FP7[idt + ii*BlockXYZi ] = 0.0;


		//  Main cycle of extrapolation 
		// FP0 + hh( a1*FP1 + a2*FP2 + a3*FP3 + a4*FP4 + a5*FP5  + a6*FP6 )
		for (int kk = kkbeg; kk <= 4; ++kk) 
		{
			ltek = l + kk * 6 - 6;
			// ����� � ������ ����� FP1
			//TPN = TP0 + dc_g[kk - 1] * hh;
			TPN = TP0 + Dcg[kk - 1] * hh;

			for(int in = 0; in < NY; in++ ) 
			{
				int itt = idt + in*BlockXYZi;
				//X0[itt] = FP0[itt] + hh * (dc_a[ltek] * FP1[itt] + dc_a[ltek + 1] * FP2[itt] + dc_a[ltek + 2] * FP3[itt] + dc_a[ltek + 3] * FP4[itt] + dc_a[ltek + 4]* FP5[itt] + dc_a[ltek + 5] *FP6[itt]);
				X0[itt] = FP0[itt] + hh * (Dca[ltek] * FP1[itt] + Dca[ltek + 1] * FP2[itt] + Dca[ltek + 2] * FP3[itt] + Dca[ltek + 3] * FP4[itt] + Dca[ltek + 4]* FP5[itt] + Dca[ltek + 5] *FP6[itt]);

				// accumulate values for error estimation
				//FP7[itt] += FP2[itt] * dc_a1[l1 + kk - 1];
				FP7[itt] += FP2[itt] * Dca1[l1 + kk - 1];
			}
			kernalFFxyz( TPN, X0, FP1, OP );

			// offset point
			for( int in = 0; in < NY; in++ )
			{
				int itt = idt + in*BlockXYZi;
				double tmp = FP1[itt];
				FP1[itt] = FP2[itt];
				FP2[itt] = FP3[itt];
				FP3[itt] = FP4[itt];
				FP4[itt] = FP5[itt];
				FP5[itt] = FP6[itt];
				FP6[itt] = tmp;
			}
		}
		//===============================================================//

		//===============================================================//
		// Main cycle of interpolation
		// X0 = FP0 + hh*( w1*FP1 + w2*FP2 + w3*FP3 + w4*FP4 + w5*FP5 + w6*FP6 )
		for (int kk = 1; kk <= 4; ++kk)
		{
			for ( int in = 0; in < NY; in++ )
			{
				int itt = idt + in*BlockXYZi;
				//X0[itt] =  FP0[itt] + hh * (dc_w[l2]*FP1[itt] + dc_w[l2 + 1]*FP2[itt] + dc_w[l2 + 2]*FP3[itt] + dc_w[l2 + 3]*FP4[itt] + dc_w[l2 + 4]*FP5[itt] + dc_w[l2 + 5]*FP6[itt] );
				X0[itt] =  FP0[itt] + hh * (Dcw[l2]*FP1[itt] + Dcw[l2 + 1]*FP2[itt] + Dcw[l2 + 2]*FP3[itt] + Dcw[l2 + 3]*FP4[itt] + Dcw[l2 + 4]*FP5[itt] + Dcw[l2 + 5]*FP6[itt] );
			}
			// new point for FP3 .... FP6
			//TPN = TP0 + dc_g[kk - 1] * hh;
			TPN = TP0 + Dcg[kk - 1] * hh;

			if( kk == 1) kernalFFxyz( TPN, X0, FP3, OP );
			if( kk == 2) kernalFFxyz( TPN, X0, FP4, OP );
			if( kk == 3) kernalFFxyz( TPN, X0, FP5, OP );
			if( kk == 4) kernalFFxyz( TPN, X0, FP6, OP );
			l2 += 6;
		}
		//===============================================================//

		//------------------------------------------------//
		if( SavePoint )
		{
			isstep++;
			// ������� ��� ������ ������� �� ������� ����� ���� 1
			int itwrite = isstep*4 + 1;

			// ������ ��������
			double w_t = TP0;
			double w_x = FP0[idt + 0*BlockXYZi ];
			double w_y = FP0[idt + 1*BlockXYZi ];
			double w_z = FP0[idt + 2*BlockXYZi ];

			s_array[ itwrite ] = w_t;
			s_array[ itwrite + 1] = w_x;
			s_array[ itwrite + 2] = w_y;
			s_array[ itwrite + 3] = w_z;
		}
		//------------------------------------------------//
	}

	//===============================================================//
	// A8: interpolatin at the destination time 
	delt = t - TP0;
	for ( int in = 0; in < NY; in++ ) 
	{
		int itt = idt + in*BlockXYZi;
		rr1 = 0.0;
		for (int j = 1; j <= 13; j += 4) 
		{
			//rr1 = delt / hh * (dc_c[j - 1] * FP3[itt] + dc_c[j] * FP4[itt] + dc_c[j + 1] * FP5[itt] + dc_c[j + 2] * FP6[itt] + rr1);
			rr1 = delt / hh * (Dcc[j - 1] * FP3[itt] + Dcc[j] * FP4[itt] + Dcc[j + 1] * FP5[itt] + Dcc[j + 2] * FP6[itt] + rr1);

			X0[itt] = FP0[itt] + rr1 * hh;
		}
	}

	//===============================================================//
	idt = blockDim.x*blockIdx.x + threadIdx.x;
	// ���������� ��� ��������������
	OP->Iparam[idt].hh = hh;
	OP->Iparam[idt].NY = NY;	
	OP->Iparam[idt].NP = NP;
	OP->Iparam[idt].IP9 = IP9;

	OP->Iparam[idt].l = l;
	OP->Iparam[idt].l1 = l1;
	OP->Iparam[idt].l2 = l2;
	OP->Iparam[idt].ltek = ltek;

	OP->Iparam[idt].kkbeg = kkbeg;
	OP->Iparam[idt].delt = delt;
	OP->Iparam[idt].rr1 = rr1;
	OP->Iparam[idt].rr2 = rr2;
	OP->Iparam[idt].rotn = rotn;
	OP->Iparam[idt].dd1 = dd1;
	OP->Iparam[idt].InvertStart = InvertStart;

	// ��������� �����, ��� ������� �����������
	OP->TP0[idt] = TP0;
	for( int it = 0; it < 6; it++ )
	{
		OP->X0[idt + it*CU_BlockXYZ] = X0[it];
		OP->FP0[idt + it*CU_BlockXYZ] = FP0[it];
		OP->FP1[idt + it*CU_BlockXYZ] = FP1[it];
		OP->FP2[idt + it*CU_BlockXYZ] = FP2[it];
		OP->FP3[idt + it*CU_BlockXYZ] = FP3[it];
		OP->FP4[idt + it*CU_BlockXYZ] = FP4[it];
		OP->FP5[idt + it*CU_BlockXYZ] = FP5[it];
		OP->FP6[idt + it*CU_BlockXYZ] = FP6[it];
		OP->FP7[idt + it*CU_BlockXYZ] = FP7[it];
	}
	//===============================================================//

	//------------------------------------------------//
	if( SavePoint )
	{
		// write numb points in array
		s_array[ 0 ] = isstep + 1;
	}
	//------------------------------------------------//
	for( int it = 0; it < 6; it++ )
		OP->X0[idt + it*CU_BlockXYZ ] = X0[it];
}
//##############################################################################//
__global__ void kernalOrbitPredict_step( gpuOrbitPoint *OP, double h, double tstart_step, double t, double e, double **d_array, bool SavePoint ) //double *indc_a, double *indc_a1, double *indc_w, double *indc_g, double *indc_c
{
	int tx = threadIdx.x;
	int idt = blockDim.x*blockIdx.x + threadIdx.x;

	__shared__ int la[5];		// = { 0,24,48,72,96 };
	__shared__ int la1[5];		// = { 0,8,16,24,32 };
	__shared__ int la2[5];		// = { 0,24,24,24,48 };
	__shared__ double alim[2];	// = { 0.0403536069, 1.0 };
	__shared__ int isinv[5];	// = { 5,2,2,2,1 };
	__shared__ int isge[5];		// = { 4,5,5,5,3 };
	__shared__ int isle[5];		// = { 4,1,1,1,3 };
	__shared__ int iseq[5];		// = { 4,2,2,2,3 };

	// ptr to orbit for this thread
	
	
	//__shared__ double dc_a[120];
	//__shared__ double dc_a1[40];
	//__shared__ double dc_w[76];
	//__shared__ double dc_g[6];
	//__shared__ double dc_c[16];

	//__shared__ double *X0;
	//__shared__ double *FP0;
	//__shared__ double *FP1;
	//__shared__ double *FP2;
	//__shared__ double *FP3;
	//__shared__ double *FP4;
	//__shared__ double *FP5;
	//__shared__ double *FP6;
	//__shared__ double *FP7;

	double* s_array;
	if( SavePoint )
		s_array = d_array[idt];

	// ������������� ��������
	if( tx == 0 )
	{
		la[0] = 0;
		la[1] = 24;
		la[2] = 48;
		la[3] = 72;
		la[4] = 96;

		la1[0] = 0;
		la1[1] = 8;
		la1[2] = 16;
		la1[3] = 24;
		la1[4] = 32;
	
		la2[0] = 0;
		la2[1] = 24;
		la2[2] = 24;
		la2[3] = 24;
		la2[4] = 48;

		alim[0] = 0.0403536069;
		alim[1] = 1.0;

		isinv[0] = 5;
		isinv[1] = 2;
		isinv[2] = 2;
		isinv[3] = 2;
		isinv[4] = 1;

		isge[0] = 4;
		isge[1] = 5;
		isge[2] = 5;
		isge[3] = 5;
		isge[4] = 3;

		isle[0] = 4;
		isle[1] = 1;
		isle[2] = 1;
		isle[3] = 1;
		isle[4] = 3;

		iseq[0] = 4;
		iseq[1] = 2;
		iseq[2] = 2;
		iseq[3] = 2;
		iseq[4] = 3;

		//X0 = OP->X0;
		//FP0 = OP->FP0;
		//FP1 = OP->FP1;
		//FP2 = OP->FP2;
		//FP3 = OP->FP3;
		//FP4 = OP->FP4;
		//FP5 = OP->FP5;
		//FP6 = OP->FP6;
		//FP7 = OP->FP7;
	}

	// �������� �������������
	//if( tx < 120 )
	//	dc_a[tx] = indc_a[tx];
	//if( tx < 40 )
	//	dc_a1[tx] = indc_a1[tx];
	//if( tx < 76 )
	//	dc_w[tx] = indc_w[tx];
	//if( tx < 6 )
	//	dc_g[tx] = indc_g[tx];
	//if( tx < 16 )
	//	dc_c[tx] = indc_c[tx];

	__syncthreads ();

	//// ��������� ����������
	//// �� ������� �����
	double X0[6];
	double FP0[6];
	double FP1[6];
	double FP2[6];
	double FP3[6];
	double FP4[6];
	double FP5[6];
	double FP6[6];
	double FP7[6];

	double Xres[6];
	
	//// ��������� ������ ��� ������� ������
	//// � ����� ����� �������� ���, ��� ��� ���� �������� � ���������� �������
	//// ����� ����������
	for( int it = 0; it < 6; it++ )
	{
		X0[it] = OP->X0[idt + it*CU_BlockXYZ];
		FP0[it] = OP->FP0[idt + it*CU_BlockXYZ];
		FP1[it] = OP->FP1[idt + it*CU_BlockXYZ];
		FP2[it] = OP->FP2[idt + it*CU_BlockXYZ];
		FP3[it] = OP->FP3[idt + it*CU_BlockXYZ];
		FP4[it] = OP->FP4[idt + it*CU_BlockXYZ];
		FP5[it] = OP->FP5[idt + it*CU_BlockXYZ];
		FP6[it] = OP->FP6[idt + it*CU_BlockXYZ];
		FP7[it] = OP->FP7[idt + it*CU_BlockXYZ];
	}

	double TP0 = OP->TP0[idt];
	double TPN = 0;
	//===============================================================//
	// ��������� ��� ����������
	//double hh = h;
	//int NY = 6;	
	//int NP = 0;
	//int IP9 = 2;

	//int l = 0;
	//int l1 = 0;
	//int l2 = 0;
	//int ltek = 0;

	//int kkbeg = 1;
	//double delt;
	//double rr1;
	//double rr2;
	//double rotn;
	//double dd1;
	//bool InvertStart = false;

	double hh = OP->Iparam[idt].hh;
	int NY = OP->Iparam[idt].NY;	
	int NP = OP->Iparam[idt].NP;
	int IP9 = OP->Iparam[idt].IP9;

	int l = OP->Iparam[idt].l;
	int l1 = OP->Iparam[idt].l1;
	int l2 = OP->Iparam[idt].l2;
	int ltek = OP->Iparam[idt].ltek;

	int kkbeg = OP->Iparam[idt].kkbeg;
	double delt = OP->Iparam[idt].delt;
	double rr1 = OP->Iparam[idt].rr1;
	double rr2 = OP->Iparam[idt].rr2;
	double rotn = OP->Iparam[idt].rotn;
	double dd1 = OP->Iparam[idt].dd1;
	bool InvertStart = OP->Iparam[idt].InvertStart;

	// new t_steppt
	double t_steppt = tstart_step;//TP0; // ����� ������ ������ �����
	int itwrites = 0;
	double PointTStep = STEPDTIME/1000.0;
	//===============================================================//

	//!!!!! ����� ��������
	idt = 0;
	int BlockXYZi = 1;
	//===============================================================//

	//------------------------------------------------//
	//if( SavePoint )
	//{
	//	// ������ ���������� ���������
	//	int itwrite = 1;
	//	double w_t = TP0;
	//	double w_x = X0[0];
	//	double w_y = X0[1];
	//	double w_z = X0[2];
	//	s_array[ itwrite ] = w_t;
	//	s_array[ itwrite + 1] = w_x;
	//	s_array[ itwrite + 2] = w_y;
	//	s_array[ itwrite + 3] = w_z;
	//}
	//------------------------------------------------//

	if (hh * (t - TP0) < 0.0)
	{ 
		//printf("direct change\n");
		// direct change.

		for( int it = 0; it < NY; it++ )
		{
			int itt = idt + it*BlockXYZi;

			double tmp = FP1[itt];
			FP1[itt] = FP6[itt];
			FP6[itt] = tmp;

			tmp = FP2[itt];
			FP2[itt] = FP5[itt];
			FP5[itt] = tmp;

			tmp = FP3[itt];
			FP3[itt] = FP4[itt];
			FP4[itt] = tmp;
		}

		// step change
		hh = -hh;
		if (IP9 == 1) {	hh /= 0.7;	}
		if (IP9 == 5) {	hh *= 0.7;	}

		// case nuber cange 
		IP9 = isinv[IP9 - 1];

		//  2 steps of extrapolation
		kkbeg = 3;

		//goto L1400;
		InvertStart = true;
	}
	//===============================================================//

	//int isstep = 0;
	// main cyrcle
	while( 1 )
	{
		if( InvertStart == false )
		{
			//===============================================================//
			//  S4: time overflow 
			if (hh * (t - TP0 - hh) <= 0.0 )
				break; 

			//+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++//
			// Add new 
			if( SavePoint )
			{
				while(hh * ( t_steppt - TP0 - hh) <= 0.0 )
				{
					// A8: interpolatin at the destination time 
					double delts = t_steppt - TP0;
					
					//for ( int ii = 0; ii < NY; ii++ ) 
					//{
					//	rr1 = 0.0;
					//	for (int j = 1; j <= 13; j += 4) 
					//	{
					//		rr1 = delts / hh * (c__[j - 1] * FP3[ii] + c__[j] * FP4[ii] + c__[j + 1] * FP5[ii] + c__[j + 2] * FP6[ii] + rr1);
					//		Xres[ii] = FP0[ii] + rr1 * hh;
					//	}
					//}

					for ( int in = 0; in < NY; in++ ) 
					{
						int itt = idt + in*BlockXYZi;
						rr1 = 0.0;
						for (int j = 1; j <= 13; j += 4) 
						{
							//rr1 = delt / hh * (dc_c[j - 1] * FP3[itt] + dc_c[j] * FP4[itt] + dc_c[j + 1] * FP5[itt] + dc_c[j + 2] * FP6[itt] + rr1);
							rr1 = delts / hh * (Dcc[j - 1] * FP3[itt] + Dcc[j] * FP4[itt] + Dcc[j + 1] * FP5[itt] + Dcc[j + 2] * FP6[itt] + rr1);
							Xres[in] = FP0[itt] + rr1 * hh;
						}
					}
					//FILE *fre = fopen( "pt.log", "at" );
					//for ( int ii = 0; ii < NY; ii++ ) 
					//	fprintf( fre, "%f\t", Xres[ii] );
					//fprintf( fre, "%f\n", t_steppt );
					//fclose( fre );

					int iw = itwrites*4 + 1;
					s_array[ iw  ] = t_steppt;
					s_array[ iw  + 1] = Xres[0];
					s_array[ iw  + 2] = Xres[1];
					s_array[ iw  + 3] = Xres[2];
					itwrites++;
					
					// next pt
					t_steppt +=  PointTStep;
				}
			}
			//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++//
			
			//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
			//if( isstep > 2990 )
			//	break;

			// if overflow absent => step of implicit RK-method 
			// A5: step of implicit RK-method + error estimation
			TP0 +=hh;

			// DELT = maximum error
			// RR1,RR2 - working values
			// ROTH - error for i-th equation
			delt = 0.0;
			for (int in = 0; in < NY; in++ )
			{
				int itt = idt + in*BlockXYZi;
				//rr2 = FP3[itt]*dc_w[72] + FP4[itt]*dc_w[73] + FP5[itt]*dc_w[74] + FP6[itt]*dc_w[75];
				rr2 = FP3[itt]*Dcw[72] + FP4[itt]*Dcw[73] + FP5[itt]*Dcw[74] + FP6[itt]*Dcw[75];

				//step counter starting setting NP = 0;
				if (NP == 0 || kkbeg == 3 ){	
					FP0[itt] +=  hh * rr2;
					continue;
				}

				//rr1 = FP7[itt] + FP2[itt]*dc_a1[l1 + 4] + FP3[itt]*dc_a1[l1 + 5] + FP4[itt]*dc_a1[l1 + 6] + FP5[itt]*dc_a1[l1 + 7];
				rr1 = FP7[itt] + FP2[itt]*Dca1[l1 + 4] + FP3[itt]*Dca1[l1 + 5] + FP4[itt]*Dca1[l1 + 6] + FP5[itt]*Dca1[l1 + 7];
				dd1 = hh * (rr1 - rr2) / e;
				rotn = fabs(dd1);
				if (rotn >= delt){	delt = rotn;}

				FP0[itt] +=  hh * rr2;
			}
			NP++;
			//===============================================================//

			//===============================================================//
			//STEP: increase, decrease or do not change? 
			if ( NP == 1 || kkbeg == 3 ){ 
				IP9 = iseq[IP9 - 1];
				kkbeg = 1;
			}
			else if(alim[0] >= delt){	
				// A6:step increase 
				if (IP9 == 1) {	hh = hh; }
				if (IP9 == 2) {	hh /= 0.7;	}
				if (IP9 == 3) {	hh /= 0.7;	}
				if (IP9 == 4) {	hh /= 0.7;	}
				if (IP9 == 5) {	hh = hh;	}
				IP9 = isge[IP9 - 1];
			}
			else if (alim[1] <= delt) {	
				// A7:step decrease 
				if (IP9 == 1) {	hh = hh; }
				if (IP9 == 2) {	hh *= 0.7;	}
				if (IP9 == 3) {	hh *= 0.7;	}
				if (IP9 == 4) {	hh *= 0.7;	}
				if (IP9 == 5) {	hh = hh;	}
				IP9 = isle[IP9 - 1];
			}
			else
			{
				IP9 = iseq[IP9 - 1];
				kkbeg = 1;
			}
			//===============================================================//
		}

		InvertStart = false;
		//===============================================================//
		// A4: extrapolation 2|4 points + interpolation 
		l = la[IP9 - 1];
		l1 = la1[IP9 - 1];
		l2 = la2[IP9 - 1];
		//===============================================================//

		//===============================================================//
		for(int ii = 0; ii < NY; ii++ ) 
			FP7[idt + ii*BlockXYZi ] = 0.0;


		//  Main cycle of extrapolation 
		// FP0 + hh( a1*FP1 + a2*FP2 + a3*FP3 + a4*FP4 + a5*FP5  + a6*FP6 )
		for (int kk = kkbeg; kk <= 4; ++kk) 
		{
			ltek = l + kk * 6 - 6;
			// ����� � ������ ����� FP1
			//TPN = TP0 + dc_g[kk - 1] * hh;
			TPN = TP0 + Dcg[kk - 1] * hh;

			for(int in = 0; in < NY; in++ ) 
			{
				int itt = idt + in*BlockXYZi;
				//X0[itt] = FP0[itt] + hh * (dc_a[ltek] * FP1[itt] + dc_a[ltek + 1] * FP2[itt] + dc_a[ltek + 2] * FP3[itt] + dc_a[ltek + 3] * FP4[itt] + dc_a[ltek + 4]* FP5[itt] + dc_a[ltek + 5] *FP6[itt]);
				X0[itt] = FP0[itt] + hh * (Dca[ltek] * FP1[itt] + Dca[ltek + 1] * FP2[itt] + Dca[ltek + 2] * FP3[itt] + Dca[ltek + 3] * FP4[itt] + Dca[ltek + 4]* FP5[itt] + Dca[ltek + 5] *FP6[itt]);

				// accumulate values for error estimation
				//FP7[itt] += FP2[itt] * dc_a1[l1 + kk - 1];
				FP7[itt] += FP2[itt] * Dca1[l1 + kk - 1];
			}
			kernalFFxyz( TPN, X0, FP1, OP );

			// offset point
			for( int in = 0; in < NY; in++ )
			{
				int itt = idt + in*BlockXYZi;
				double tmp = FP1[itt];
				FP1[itt] = FP2[itt];
				FP2[itt] = FP3[itt];
				FP3[itt] = FP4[itt];
				FP4[itt] = FP5[itt];
				FP5[itt] = FP6[itt];
				FP6[itt] = tmp;
			}
		}
		//===============================================================//

		//===============================================================//
		// Main cycle of interpolation
		// X0 = FP0 + hh*( w1*FP1 + w2*FP2 + w3*FP3 + w4*FP4 + w5*FP5 + w6*FP6 )
		for (int kk = 1; kk <= 4; ++kk)
		{
			for ( int in = 0; in < NY; in++ )
			{
				int itt = idt + in*BlockXYZi;
				//X0[itt] =  FP0[itt] + hh * (dc_w[l2]*FP1[itt] + dc_w[l2 + 1]*FP2[itt] + dc_w[l2 + 2]*FP3[itt] + dc_w[l2 + 3]*FP4[itt] + dc_w[l2 + 4]*FP5[itt] + dc_w[l2 + 5]*FP6[itt] );
				X0[itt] =  FP0[itt] + hh * (Dcw[l2]*FP1[itt] + Dcw[l2 + 1]*FP2[itt] + Dcw[l2 + 2]*FP3[itt] + Dcw[l2 + 3]*FP4[itt] + Dcw[l2 + 4]*FP5[itt] + Dcw[l2 + 5]*FP6[itt] );
			}
			// new point for FP3 .... FP6
			//TPN = TP0 + dc_g[kk - 1] * hh;
			TPN = TP0 + Dcg[kk - 1] * hh;

			if( kk == 1) kernalFFxyz( TPN, X0, FP3, OP );
			if( kk == 2) kernalFFxyz( TPN, X0, FP4, OP );
			if( kk == 3) kernalFFxyz( TPN, X0, FP5, OP );
			if( kk == 4) kernalFFxyz( TPN, X0, FP6, OP );
			l2 += 6;
		}
		//===============================================================//

		//------------------------------------------------//
		//if( SavePoint )
		//{
		//	isstep++;
		//	// ������� ��� ������ ������� �� ������� ����� ���� 1
		//	int itwrite = isstep*4 + 1;

		//	// ������ ��������
		//	double w_t = TP0;
		//	double w_x = FP0[idt + 0*BlockXYZi ];
		//	double w_y = FP0[idt + 1*BlockXYZi ];
		//	double w_z = FP0[idt + 2*BlockXYZi ];

		//	s_array[ itwrite ] = w_t;
		//	s_array[ itwrite + 1] = w_x;
		//	s_array[ itwrite + 2] = w_y;
		//	s_array[ itwrite + 3] = w_z;
		//}
		//------------------------------------------------//
	}

	//===============================================================//
	// A8: interpolatin at the destination time 
	delt = t - TP0;
	for ( int in = 0; in < NY; in++ ) 
	{
		int itt = idt + in*BlockXYZi;
		rr1 = 0.0;
		for (int j = 1; j <= 13; j += 4) 
		{
			//rr1 = delt / hh * (dc_c[j - 1] * FP3[itt] + dc_c[j] * FP4[itt] + dc_c[j + 1] * FP5[itt] + dc_c[j + 2] * FP6[itt] + rr1);
			rr1 = delt / hh * (Dcc[j - 1] * FP3[itt] + Dcc[j] * FP4[itt] + Dcc[j + 1] * FP5[itt] + Dcc[j + 2] * FP6[itt] + rr1);

			X0[itt] = FP0[itt] + rr1 * hh;
		}
	}

	//+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++//
	// Add new 
	if( SavePoint )
	{
		while( hh*( t_steppt - t) <= 0.0 )
		{
			// A8: interpolatin at the destination time 
			double delts = t_steppt - TP0;

			//for ( int ii = 0; ii < NY; ii++ ) 
			//{
			//	rr1 = 0.0;
			//	for (int j = 1; j <= 13; j += 4) 
			//	{
			//		rr1 = delts / hh * (c__[j - 1] * FP3[ii] + c__[j] * FP4[ii] + c__[j + 1] * FP5[ii] + c__[j + 2] * FP6[ii] + rr1);
			//		Xres[ii] = FP0[ii] + rr1 * hh;
			//	}
			//}

			for ( int in = 0; in < NY; in++ ) 
			{
				int itt = idt + in*BlockXYZi;
				rr1 = 0.0;
				for (int j = 1; j <= 13; j += 4) 
				{
					//rr1 = delt / hh * (dc_c[j - 1] * FP3[itt] + dc_c[j] * FP4[itt] + dc_c[j + 1] * FP5[itt] + dc_c[j + 2] * FP6[itt] + rr1);
					rr1 = delts / hh * (Dcc[j - 1] * FP3[itt] + Dcc[j] * FP4[itt] + Dcc[j + 1] * FP5[itt] + Dcc[j + 2] * FP6[itt] + rr1);

					Xres[in] = FP0[itt] + rr1 * hh;
				}
			}

			//FILE *fre = fopen( "pt.log", "at" );
			//for ( int ii = 0; ii < NY; ii++ ) 
			//	fprintf( fre, "%f\t", Xres[ii] );
			//fprintf( fre, "%f\n", t_steppt );
			//fclose( fre );

			int iw = itwrites*4 + 1;
			s_array[ iw ] = t_steppt;
			s_array[ iw + 1] = Xres[0];
			s_array[ iw + 2] = Xres[1];
			s_array[ iw + 3] = Xres[2];
			itwrites++;

			// next pt
			t_steppt += PointTStep;
		}
	}
	//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++//
	__syncthreads ();

	//===============================================================//
	idt = blockDim.x*blockIdx.x + threadIdx.x;
	// ���������� ��� ��������������
	OP->Iparam[idt].hh = hh;
	OP->Iparam[idt].NY = NY;	
	OP->Iparam[idt].NP = NP;
	OP->Iparam[idt].IP9 = IP9;

	OP->Iparam[idt].l = l;
	OP->Iparam[idt].l1 = l1;
	OP->Iparam[idt].l2 = l2;
	OP->Iparam[idt].ltek = ltek;

	OP->Iparam[idt].kkbeg = kkbeg;
	OP->Iparam[idt].delt = delt;
	OP->Iparam[idt].rr1 = rr1;
	OP->Iparam[idt].rr2 = rr2;
	OP->Iparam[idt].rotn = rotn;
	OP->Iparam[idt].dd1 = dd1;
	OP->Iparam[idt].InvertStart = InvertStart;

	// ��������� �����, ��� ������� �����������
	OP->TP0[idt] = TP0;
	for( int it = 0; it < 6; it++ )
	{
		OP->X0[idt + it*CU_BlockXYZ] = X0[it];
		OP->FP0[idt + it*CU_BlockXYZ] = FP0[it];
		OP->FP1[idt + it*CU_BlockXYZ] = FP1[it];
		OP->FP2[idt + it*CU_BlockXYZ] = FP2[it];
		OP->FP3[idt + it*CU_BlockXYZ] = FP3[it];
		OP->FP4[idt + it*CU_BlockXYZ] = FP4[it];
		OP->FP5[idt + it*CU_BlockXYZ] = FP5[it];
		OP->FP6[idt + it*CU_BlockXYZ] = FP6[it];
		OP->FP7[idt + it*CU_BlockXYZ] = FP7[it];
	}
	//===============================================================//

	//------------------------------------------------//
	if( SavePoint )
	{
		// write numb points in array
		s_array[ 0 ] = itwrites;
	}
	//------------------------------------------------//
	for( int it = 0; it < 6; it++ )
		OP->X0[idt + it*CU_BlockXYZ ] = X0[it];
}
//##############################################################################//

//==============================================================================//
// ������� ������������ �� ��������
//==============================================================================//
__device__ double cuGetInterpolatePoint( double *x, double *y, int np, double xf )
{
	double res = 0.0;
	// ��������� ������� ��������
	for( int it = 0; it < np; it++ )
	{
		double tmp = 1.0;
		// ��������� �������
		for( int j = 0; j < np; j++ )
		{
			// ���������� ���������
			if( j == it )
				j++; 

			tmp = tmp*(xf-x[j])/(x[it]-x[j]);
		}

		// �������� �� �������� � ���� �����
		tmp = tmp*y[it];

		// ���������
		res = res + tmp;
	}
	return res;
};
//==============================================================================//
// ����� ���������� ����� �������
//==============================================================================//
__device__ double cuGetDistTwoPoints( double x1, double y1, double z1, double x2, double y2, double z2 )
{
	double dist = sqrt( (x2-x1)*(x2-x1) + (y2-y1)*(y2-y1) + (z2-z1)*(z2-z1) );
	return dist;
}
//==============================================================================//
// �������� ������� ���������
//==============================================================================//
__global__ void cuFindSmallDistant( double **d_array, double *ResultFindMinDist, int *NumbResult, int size )
{
	// ����� ������
	int ns = blockDim.x*blockIdx.x + threadIdx.x;
	if( ns < size-1 )
	{
		// ��������� �� ������
		double *OrbitPointsArray_S0 = d_array[0];
		double *OrbitPointsArray_S1 = d_array[ns+1];

		// ��������� ������
		// ����� ����� ��� ������������
		const int Nlograng = 5;
		const int Nlograng_half = 2;
		// ������� ��� ������, � ������� �����
		int Ns = 3;
		// ��������� �� �����
		int Ne = 3;
		// ��� �� ������� � ��� ��������
		double dT = 1.0/1000.0;	
		double minDist = 10000000000.0;
		double minDistTime = 0;

		double Ts = OrbitPointsArray_S0[ Ns*4+1 ];		// ����� - ����� ������, ��������� �� ������
		int Np0 = (int)OrbitPointsArray_S0[0];				// ����� �����
		int Np1 = (int)OrbitPointsArray_S1[0];				// ����� �����
		double Te0 = OrbitPointsArray_S0[ (Np0-1-Ne)*4+1 ];	// ����� ���������, ��������� �� �����
		double Te1 = OrbitPointsArray_S1[ (Np1-1-Ne)*4+1 ];	// ����� ���������, ��������� �� �����
		// �������� ����������� �����, ����� �� ������� �� ������� ������� �����
		double Te;
		if( Te0 < Te1 )
			Te = Te0;
		else 
			Te = Te1;

		// ���������
		//printf("%d\t %d\t %d\t %f\t %f\t %f\n", ns, Np0, Np1, Ts, Te, dT );

		// ������� ������� - ������� � ������� ������
		int crN0 = Ns;  // ��� ����� �� �����
		int crN1 = 1;	// ��� ����� ����� ��������
		
		// ��� ������ ��������
		double LastDist;
		int Direction = 0;
		int iterFind = 0;
		PointDist pt1; // �������
		PointDist pt2; // �������
		PointDist pt3; // ��� ���� �����

		pt1.d = 0;
		pt1.t = 0;
		pt2.d = 0;
		pt2.t = 0;
		pt3.d = 0;
		pt3.t = 0;

		__syncthreads ();
		// ���� �� ������� � �����
		for( double iT = Ts; iT < Te; iT += dT )
		{
			// iT - ������� �����
			// �� ������ ��������� � ������ ������
			// �������� �� ����� �� �� �������
			if( crN0 >= Np0-1-Ne || crN1 >= Np1-1-Ne )
				break;

			// ��������� ����� ����� ������ �������
			while( 1 ){
				// ���������� ��������� ����� � �������
				if( iT > OrbitPointsArray_S0[ (crN0-1)*4+1 ] && iT < OrbitPointsArray_S0[ (crN0+1)*4+1 ] )	break;
				// �������� ������� ������ �������, ��������� �������
				if( iT <= OrbitPointsArray_S0[ (crN0-1)*4+1 ] )	crN0--;
				// �������� ������� ������ ������� ��������, ����������� �������
				if( iT >= OrbitPointsArray_S0[ (crN0+1)*4+1 ] )	crN0++;
			}
			while( 1 )	{
				// ���������� ��������� ����� � �������
				if( iT > OrbitPointsArray_S1[ (crN1-1)*4+1 ] && iT < OrbitPointsArray_S1[ (crN1+1)*4+1 ] )	break;
				// �������� ������� ������ �������, ��������� �������
				if( iT <= OrbitPointsArray_S1[ (crN1-1)*4+1 ] )	crN1--;
				// �������� ������� ������ ������� ��������, ����������� �������
				if( iT >= OrbitPointsArray_S1[ (crN1+1)*4+1 ] )	crN1++;
			}

			// �������� �� ����� �� �� �������
			if( crN0 >= Np0-1-Ne || crN1 >= Np1-1-Ne )
				break;

			if( crN0 <= Nlograng_half || crN1 <= Nlograng_half )
				continue;
			// ������ ��� ������������
			// crN0, crN1 - ������� ����������� �����
			// iT - �����
			// �������� ����� ��� ������������, ����� � ��� �� �� ��������
					// �������
			double PT0[Nlograng];
			double SX0[Nlograng];
			double SY0[Nlograng];
			double SZ0[Nlograng];
			int k = 0;
			for( int it = crN0 - Nlograng_half; it <= crN0 + Nlograng_half; it++)
			{
				PT0[k] = OrbitPointsArray_S0[ it*4+1 ];
				SX0[k] = OrbitPointsArray_S0[ it*4+1+1 ];
				SY0[k] = OrbitPointsArray_S0[ it*4+1+2 ];
				SZ0[k] = OrbitPointsArray_S0[ it*4+1+3 ];
				k++;
			}
			// ����� �������� ����� � �������������
			double x0 = cuGetInterpolatePoint( PT0, SX0, Nlograng, iT );
			double y0 = cuGetInterpolatePoint( PT0, SY0, Nlograng, iT );
			double z0 = cuGetInterpolatePoint( PT0, SZ0, Nlograng, iT );

			k = 0;
			for( int it = crN1 - Nlograng_half; it <= crN1 + Nlograng_half; it++)
			{
				PT0[k] = OrbitPointsArray_S1[ it*4+1 ];
				SX0[k] = OrbitPointsArray_S1[ it*4+1+1 ];
				SY0[k] = OrbitPointsArray_S1[ it*4+1+2 ];
				SZ0[k] = OrbitPointsArray_S1[ it*4+1+3 ];
				k++;
			}
			double x1 = cuGetInterpolatePoint( PT0, SX0, Nlograng, iT );
			double y1 = cuGetInterpolatePoint( PT0, SY0, Nlograng, iT );
			double z1 = cuGetInterpolatePoint( PT0, SZ0, Nlograng, iT );

			// �������� ���������� ����� �������
			double d =  cuGetDistTwoPoints( x0, y0, z0, x1, y1, z1 );
			d = d*1000.0; // ��

			if( d < minDist )
			{
				minDist = d;
				minDistTime = iT;
			}

			//==========================================//
			// ���� ����������� ��������
			bool flagReverse = false;
			if( iterFind > 0 )
			{
				// ����������� �� ���������� �������������
				int nowDirection;
				if( d < LastDist )
					nowDirection = 1;
				else
					nowDirection = -1;

				// ����������� ����������
				if( nowDirection < 0 && Direction > 0 )
				{
					flagReverse = true;
				}

				// ��������� �����������
				Direction = nowDirection;
			}
			// ��������� ���������� �������
			LastDist = d;

			// ����� �����
			pt3 = pt2;
			pt2 = pt1;
			// ����� ��������� ����� ����������� �����������
			pt1.t = iT;
			pt1.d = d;

			// ���� ���� ��������� �����������
			// ������������� ����������
			double InterpolateDist = 100000000000;
			double InterpolateTime = 0;
			if( flagReverse == true && iterFind > 2 )
			{
				double x1 = pt3.t;
				double x2 = pt2.t;
				double x3 = pt1.t;

				double y1 = pt3.d;
				double y2 = pt2.d;
				double y3 = pt1.d;

				double a = y3 - ( x3*(y2-y1) + x2*y1 - x1*y2 )/(x2 - x1);
				a = a/( x3*( x3 - x1 - x2 ) + x1*x2 );

				double b = (y2 - y1)/(x2 - x1) - a*( x1 + x2 );

				double c = (x2*y1 - x1*y2)/(x2 - x1) + a*x1*x2;

				double discr = b*b - 4.0*a*c;

				double yv = 0;
				double xv = 0;
				if( a!= 0 )
				{
					yv = -discr/4.0/a;
					xv = -b/2.0/a;
				}
				
				InterpolateDist = yv;
				InterpolateTime = xv;
			}
			//==========================================//
			// ��������� ����
			if( d >= 100 && d < 300 )
				dT = 1.0/1000.0;
			else if( d < 100 )
				dT = 0.1/1000.0;
			else
				dT = 5.0/1000.0;

			if( flagReverse == true && InterpolateDist < 100 )
			{
				int npos = atomicAdd( &NumbResult[0], 1 );
				//int npos = (int)ResultFindMinDist[0];

				int indexpos = 3*npos + 1;
				ResultFindMinDist[ indexpos ] = ns+1;
				ResultFindMinDist[ indexpos + 1] = InterpolateTime;
				ResultFindMinDist[ indexpos + 2 ] = InterpolateDist;
			}
			iterFind ++;
		}
	}
};
//=====================================================================//

//#####################################################################//
//						������� �������� �� GPU
//#####################################################################//
	//==============================================================================//
	// ������������� ����������� �� GPU
	//==============================================================================//
	void PredictOrbitSat::InitIntegrationGPU()
	{
		printf("Run Init Integration GPU 1\n");
		// constant memory for constant
		const double *hc_a = IC_a;
		const double *hc_a1 = IC_a1;
		const double *hc_w = IC_w;
		const double *hc_g = IC_g;
		const double *hc_c = IC_c;
		// Global memory for constant
		//double *dc_a;
		//double *dc_a1;
		//double *dc_w;
		//double *dc_g;
		//double *dc_c;
		//dc_a[120]; dc_a1[40]; dc_w[76]; dc_g[6]; dc_c[16];
		//cutilSafeCall( hipMalloc( (void**)&dc_a, sizeof(double)*120 ));
		//cutilSafeCall( hipMalloc( (void**)&dc_a1,sizeof(double)*40 ));
		//cutilSafeCall( hipMalloc( (void**)&dc_w, sizeof(double)*76 ));
		//cutilSafeCall( hipMalloc( (void**)&dc_g, sizeof(double)*6 ));
		//cutilSafeCall( hipMalloc( (void**)&dc_c, sizeof(double)*16 ));
		//cutilSafeCall( hipMemcpy( dc_a, hc_a, sizeof(double)*120, hipMemcpyHostToDevice ));
		//cutilSafeCall( hipMemcpy( dc_a1, hc_a1, sizeof(double)*40, hipMemcpyHostToDevice ));
		//cutilSafeCall( hipMemcpy( dc_w, hc_w, sizeof(double)*76, hipMemcpyHostToDevice ));
		//cutilSafeCall( hipMemcpy( dc_g, hc_g, sizeof(double)*6, hipMemcpyHostToDevice ));
		//cutilSafeCall( hipMemcpy( dc_c, hc_c, sizeof(double)*16, hipMemcpyHostToDevice ));

		int inkkr[12] = { 3,4,5,6,1,2,4,3,2,1,6,5 };
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( KKR), inkkr, sizeof(int)*12, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( Dca), hc_a, sizeof(double)*120, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( Dca1), hc_a1, sizeof(double)*40, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( Dcw), hc_w, sizeof(double)*76, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( Dcg), hc_g, sizeof(double)*6, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL( Dcc), hc_c, sizeof(double)*16, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUtick), hh_tick, sizeof(double)*42, 0, hipMemcpyHostToDevice ));

		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUaal0), h_aal0, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUaal1), h_aal1, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUaal2), h_aal2, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUaal3), h_aal3, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUaal4), h_aal4, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUaa0), h_aa0, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUaa1), h_aa1, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUaa2), h_aa2, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUaa3), h_aa3, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUaa4), h_aa4, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUaa5), h_aa5, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUaa6), h_aa6, sizeof(double)*14, 0, hipMemcpyHostToDevice ));

		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUbb0), h_bb0, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUbb1), h_bb1, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUbb2), h_bb2, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUbb3), h_bb3, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUbb4), h_bb4, sizeof(double)*14, 0, hipMemcpyHostToDevice ));

		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUcc0), h_cc0, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUcc1), h_cc1, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUcc2), h_cc2, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUcc3), h_cc3, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUcc4), h_cc4, sizeof(double)*14, 0, hipMemcpyHostToDevice ));

		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUdd0), h_dd0, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUdd1), h_dd1, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUdd2), h_dd2, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUdd3), h_dd3, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUdd4), h_dd4, sizeof(double)*14, 0, hipMemcpyHostToDevice ));

		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUee0), h_ee0, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUee1), h_ee1, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUee2), h_ee2, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUee3), h_ee3, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUee4), h_ee4, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUee5), h_ee5, sizeof(double)*7, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUee6), h_ee6, sizeof(double)*7, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUee7), h_ee7, sizeof(double)*7, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUee8), h_ee8, sizeof(double)*7, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUff1), h_ff1, sizeof(double)*7, 0, hipMemcpyHostToDevice ));

		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUeet5), h_eet5, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUeet6), h_eet6, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUeet7), h_eet7, sizeof(double)*14, 0, hipMemcpyHostToDevice ));
		cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(  CUeet8), h_eet8, sizeof(double)*14, 0, hipMemcpyHostToDevice ));

		hipEventCreate ( &start );
		hipEventCreate ( &stop );

		printf("Init Integration GPU OK\n");
	}
	//==============================================================================//
	// delete memory
	//==============================================================================//
	void PredictOrbitSat::DeleteIntegrationGPU()
	{
		// no Create momory today
		hipEventDestroy ( start );
		hipEventDestroy ( stop );
	};
	//==============================================================================//
	// ���������� �������� ��� ������
	//==============================================================================//
	void PredictOrbitSat::RunIntegrationGpu( double t_s, double t, double h, double e, SatelliteArray &ListSat, SatelliteArray &ListSatResGpu )
	{
		//---------------------------------------------//
		hipEventRecord ( start, 0 );
		// ������� ����
		dim3 threadsRK( CU_TableSize );
		dim3 gridRK( CU_GridSize );
		// ��������� ��������
		// ��������� �� ��������� �� ����������
		// ����������� ���������
		gpuOrbitPoint d_OP;
		d_OP.AllocMemory( 6, CU_BlockXYZ );
		d_OP.CopyToGPU( ListSat );
		// ������� ����������
		d_OP.d_EF = IF->d_FileMemDE403;
		d_OP.dNUT_AMPL =IF->d_AMPL;
		d_OP.dNUT_ARG = IF->d_ARG;
		d_OP.dT_finals_n = IF->d_finals_n;
		d_OP.dT_finals_tab = IF->d_finals_tab;
		d_OP.d_Garmonic = IF->d_egm96;

		d_OP.ajd0 = IF->S_ajd0;
		d_OP.delt0 = IF->S_delt0;
		d_OP.Satm = IF->SIGMA_ATM;
		d_OP.Ssun = IF->SIGMA_SUN;

		// �����������
		gpuOrbitPoint *dpointer_OP;
		cutilSafeCall( hipMalloc( (void**)&dpointer_OP, sizeof( gpuOrbitPoint ) ) );
		cutilSafeCall( hipMemcpy( dpointer_OP, &d_OP, sizeof( gpuOrbitPoint ), hipMemcpyHostToDevice) );

		hipEventRecord ( stop, 0 );
		hipEventSynchronize ( stop );
		hipEventElapsedTime ( &gpuTime, start, stop );
		printf("GPU prepare Time =  %.2f ms\n", gpuTime );
		//---------------------------------------------//

		hipEventRecord ( start, 0 );
		
		kernalOrbitInitParam<<< gridRK, threadsRK>>>( dpointer_OP, h );
		kernalGetStartPoint<<< gridRK, threadsRK>>>( dpointer_OP, h );
		kernalOrbitPredict<<< gridRK, threadsRK>>>( dpointer_OP, h, t, e, NULL, false );

		hipDeviceSynchronize();
		hipEventRecord ( stop, 0 );
		hipEventSynchronize ( stop );
		hipEventElapsedTime ( &gpuTime, start, stop );
		printf("GPU cuGetStartPoint and OrbitPredict Time =  %.2f ms\n", gpuTime );

		// From GPU
		d_OP.CopyFromGPU( ListSatResGpu );
		d_OP.DeleteMemory();
		return;
	};
	//==============================================================================//
	// ���������� ������ � ����� ������� ���������
	// t_s - ����� ������
	// t - ����� �����
	// h - ���
	// � - ������ ��������������
	// ListSat - �������
	// ListSatVerify - ������ ��� ��������
	//==============================================================================//
	void PredictOrbitSat::RunIntegrationGpu_Approach( double t_s, double t, double h, double e, SatelliteArray &ListSat, SatelliteArray &ListSatVerify, std::vector< PointWithMinDist > &result )
	{
		printf("GPU Version (Big Step Time)\n"); 

		//---------------------------------------------//
		hipEventRecord ( start, 0 );
		// ������� ����
		dim3 threadsRK( CU_TableSize );
		dim3 gridRK( CU_GridSize );

		// ��������� ��������
		// ��������� �� ��������� �� ����������
		// ����������� ���������
		gpuOrbitPoint d_OP;
		d_OP.AllocMemory( 6, CU_BlockXYZ );
		d_OP.CopyToGPU( ListSat );

		// ������� ����������
		d_OP.d_EF = IF->d_FileMemDE403;
		d_OP.dNUT_AMPL =IF->d_AMPL;
		d_OP.dNUT_ARG = IF->d_ARG;
		d_OP.dT_finals_n = IF->d_finals_n;
		d_OP.dT_finals_tab = IF->d_finals_tab;
		d_OP.d_Garmonic = IF->d_egm96;

		d_OP.ajd0 = IF->S_ajd0;
		d_OP.delt0 = IF->S_delt0;
		d_OP.Satm = IF->SIGMA_ATM;
		d_OP.Ssun = IF->SIGMA_SUN;

		// ����������� �� gpu
		gpuOrbitPoint *dpointer_OP;
		cutilSafeCall( hipMalloc( (void**)&dpointer_OP, sizeof( gpuOrbitPoint ) ) );
		cutilSafeCall( hipMemcpy( dpointer_OP, &d_OP, sizeof( gpuOrbitPoint ), hipMemcpyHostToDevice) );

		// buffer constant
		double NumbdT = 8.0;
		const double TimeStep = 86.4/NumbdT;			// ��� ������� ����� ������������
		const int MaxSizePoints = 21600 / NumbdT + 10;		//21600	// ������������ ����� ����� �� ������17280

		// �������� ������
		double dtt = t - t_s;
		// ����� ����� �� ��������
		int ntt = (int)(dtt/TimeStep);
		printf( "%f %d %f %f\n", dtt, ntt, t_s, t );

		// ������ � ������������ �������
		double *TPR = new double[ntt+1];
		double *TPR_s = new double[ntt+1];
		for( int it = 0; it < ntt; it++ )
		{
			TPR[it] = t_s + ((float)it+1)*TimeStep;
			TPR_s[it] = t_s + ((float)it)*TimeStep;
		}

		// init orbit arr
		OrbitArrayPointsGPU LAD;
		LAD.InitArrayList( CU_BlockXYZ, MaxSizePoints );
		
		OrbitArrayPointsCPU LAC;
		LAC.InitArrayList( CU_BlockXYZ, MaxSizePoints );
		
		hipEventRecord ( stop, 0 );
		hipEventSynchronize ( stop );
		hipEventElapsedTime ( &gpuTime, start, stop );
		printf("GPU prepare Time =  %.2f ms\n", gpuTime );
		//---------------------------------------------//

		//---------------------------------------------//
		// ���������� ���������� � ���������� �����
		kernalOrbitInitParam<<< gridRK, threadsRK>>>( dpointer_OP, h );
		kernalGetStartPoint<<< gridRK, threadsRK>>>( dpointer_OP, h );
		for( int it = 0; it < ntt; it++ )
		{
			hipEventRecord ( start, 0 );
			kernalOrbitPredict_step<<< gridRK, threadsRK>>>( dpointer_OP, h, TPR_s[it], TPR[it], e, LAD.d_array_list, true );
			LAD.CopyFromGPU( LAC );
			hipDeviceSynchronize();
			hipEventRecord ( stop, 0 );
			hipEventSynchronize ( stop );
			hipEventElapsedTime ( &gpuTime, start, stop );
			printf("GPU cuGetStartPoint and OrbitPredict Time =  %.2f ms\n", gpuTime );

			int dt1 = clock();

			for( int ig = 1; ig < CU_BlockXYZ; ig++ )
			{
				FindSmallDistant_cpu( LAC.array_list[0], LAC.array_list[ig], ig, MaxSizePoints, result );

				//FILE *fres = fopen( "gpures.log", "w" );
				//int n = LAC.array_list[ig][ 0 ];
				//fprintf(fres, "%d\t", n );
				//for( int di = 0; di < n; di++ )
				//	fprintf( fres, "%f ", LAC.array_list[ig][ di*4+1 ] );
				//fprintf(fres, "\n");
				//fclose( fres );
			}

			for (int it = 0; it < result.size(); it++)
			{
				int iarr = result[it].Nlist;
				int idn = result[it].norad;
				double d = result[it].d;
				double dv = 0;// result[it].d_verify;
				double t = result[it].t;
				double dd = 0;// d - dv;
				printf("%d\t %d\t %f\t %f\t %f\t %.12f\n", iarr, idn, t, d, dv, dd);
			}

			int dt2 = clock();
			printf("TIME ALL GPU Find Close Approach %f  ms\n", (double)(dt2-dt1)/CLOCKS_PER_SEC*1000.0 );
		}
		//---------------------------------------------//

		
		// From GPU
		d_OP.DeleteMemory();
		LAD.FreeArrayList();
		LAC.FreeArrayList();
		delete TPR;
		return;
	};

	/*void PredictOrbitSat::RunIntegrationGpu( double t_s, double t, double h, double e, Orbit::OrbitArrayPointsCPU &outarr, SatelliteArray &ListSat, SatelliteArray &ListSatResGpu )
	{
		printf("New Version GPU mod start .....\n"); 
	
		hipEventRecord ( start, 0 );
		// ������� ����
		dim3 threadsRK( CU_TableSize );
		dim3 gridRK( CU_GridSize );

		// ��������� ��������
		// ��������� �� ��������� �� ����������
		// ����������� ���������
		gpuOrbitPoint d_OP;
		d_OP.AllocMemory( 6, CU_BlockXYZ );
		d_OP.CopyToGPU( ListSat );
		// ������� ����������
		d_OP.d_EF = IF->d_FileMemDE403;
		d_OP.dNUT_AMPL =IF->d_AMPL;
		d_OP.dNUT_ARG = IF->d_ARG;
		d_OP.dT_finals_n = IF->d_finals_n;
		d_OP.dT_finals_tab = IF->d_finals_tab;
		d_OP.d_Garmonic = IF->d_egm96;
		// �����������
		gpuOrbitPoint *dpointer_OP;
		cutilSafeCall( hipMalloc( (void**)&dpointer_OP, sizeof( gpuOrbitPoint ) ) );
		cutilSafeCall( hipMemcpy( dpointer_OP, &d_OP, sizeof( gpuOrbitPoint ), hipMemcpyHostToDevice) );

		hipEventRecord ( stop, 0 );
		hipEventSynchronize ( stop );
		hipEventElapsedTime ( &gpuTime, start, stop );
		printf("GPU prepare Time =  %.2f ms\n", gpuTime );

		kernalOrbitInitParam<<< gridRK, threadsRK>>>( dpointer_OP, h );
		kernalGetStartPoint<<< gridRK, threadsRK>>>( dpointer_OP, h );

		// �������� ������
		double dtt = t - t_s;
		// ����� ����� �� ��������
		int ntt = (int)(dtt/40.0);
		printf( "%f %d %f %f\n", dtt, ntt, t_s, t );
		// ������ � ������������ �������
		double *TPR = new double[ntt+1];
		for( int it = 0; it < ntt; it++ )
			TPR[it] = t_s + (it+1)*40.0;
		TPR[ntt] = t;

		//-----------------------------------------//
		// ����� ���������
		int *h_numres = new int[1];
		int *d_numres;
		cutilSafeCall( hipMalloc( (void**)&d_numres, sizeof( int ) ) );

		h_numres[0] = 0;
		cutilSafeCall( hipMemcpy( d_numres, h_numres, sizeof( int ), hipMemcpyHostToDevice) );

		int sizeresarr = 3*10000+1;
		double *h_ResultFindMinDist = new double[sizeresarr];
		double *d_ResultFindMinDist;
		cutilSafeCall( hipMalloc( (void**)&d_ResultFindMinDist, sizeresarr*sizeof( double ) ) );
		//-----------------------------------------//

		OrbitArrayPointsGPU LAD;
		LAD.InitArrayList( CU_BlockXYZ, 10000 );

		for( int it = 0; it <= ntt; it++ )
		{

			hipEventRecord ( start, 0 );
			kernalOrbitPredict<<< gridRK, threadsRK>>>( dpointer_OP, h, TPR[it], e, LAD.d_array_list );
			hipDeviceSynchronize();
			hipEventRecord ( stop, 0 );
			hipEventSynchronize ( stop );
			hipEventElapsedTime ( &gpuTime, start, stop );
			printf("GPU cuGetStartPoint and OrbitPredict Time =  %.2f ms\n", gpuTime );

			hipEventRecord ( start, 0 );
			cuFindSmallDistant<<< gridRK, threadsRK>>>( LAD.d_array_list, d_ResultFindMinDist, d_numres, CU_BlockXYZ );
			hipDeviceSynchronize();
			hipEventRecord ( stop, 0 );
			hipEventSynchronize ( stop );
			hipEventElapsedTime ( &gpuTime, start, stop );
			printf("GPU cuFindSmallDistant Time =  %.2f ms\n", gpuTime );


			LAD.CopyFromGPU( outarr );
			// Nall, ni Ti Di, ....
			double *ResultFindMinDist = new double[3*10000+1]; 
			ResultFindMinDist[0] = 0;
				
			int dt1 = clock();
			for( int it = 1; it < CU_BlockXYZ; it++ )
				FindSmallDistant( outarr.array_list[0], outarr.array_list[it], it, ResultFindMinDist );

			int dt2 = clock();
			printf("TIME ALL GPU Find Close Approach %f  ms\n", (double)(dt2-dt1)/CLOCKS_PER_SEC*1000.0 );

			FILE *Fres = fopen( "DistInterpolate_gpu_ww.txt", "at" );
			int nf = ResultFindMinDist[0];
			for( int it = 0; it < nf; it++ )
			{
				int index = 3*it + 1;
				fprintf( Fres, "%.0f\t %f\t %f\n", ResultFindMinDist[index], ResultFindMinDist[index+1], ResultFindMinDist[index+2] );  
			}
			fclose( Fres );

			delete ResultFindMinDist;
		}
	
		//-----------------------------------------//
		// ��������� �����������
		cutilSafeCall( hipMemcpy( h_ResultFindMinDist, d_ResultFindMinDist, sizeresarr*sizeof( double ), hipMemcpyDeviceToHost) );
		cutilSafeCall( hipMemcpy( h_numres, d_numres, sizeof( int ), hipMemcpyDeviceToHost) );


		printf( "h_numres = %d\n", h_numres[0] );
		h_ResultFindMinDist[0] = h_numres[0];

		FILE *Fres = fopen( "DistInterpolate_gpu_arr.txt", "w" );
		int nf = h_ResultFindMinDist[0];
		for( int it = 0; it < nf; it++ )
		{
			int index = 3*it + 1;
			fprintf( Fres, "%.0f\t %f\t %f\n", h_ResultFindMinDist[index], h_ResultFindMinDist[index+1], h_ResultFindMinDist[index+2] );  
		}
		fclose( Fres );
		//-----------------------------------------//

		// From GPU
		LAD.FreeArrayList();
		d_OP.CopyFromGPU( ListSatResGpu );
		d_OP.DeleteMemory();
		return;
	};*/


/*
printf("New Version GPU mod start .....\n"); 
		//---------------------------------------------//
		hipEventRecord ( start, 0 );
		// ������� ����
		dim3 threadsRK( CU_TableSize );
		dim3 gridRK( CU_GridSize );
		// ��������� ��������
		// ��������� �� ��������� �� ����������
		// ����������� ���������
		gpuOrbitPoint d_OP;
		d_OP.AllocMemory( 6, CU_BlockXYZ );
		d_OP.CopyToGPU( ListSat );
		// ������� ����������
		d_OP.d_EF = IF->d_FileMemDE403;
		d_OP.dNUT_AMPL =IF->d_AMPL;
		d_OP.dNUT_ARG = IF->d_ARG;
		d_OP.dT_finals_n = IF->d_finals_n;
		d_OP.dT_finals_tab = IF->d_finals_tab;
		d_OP.d_Garmonic = IF->d_egm96;

		d_OP.ajd0 = IF->S_ajd0;
		d_OP.delt0 = IF->S_delt0;
		d_OP.Satm = IF->SIGMA_ATM;
		d_OP.Ssun = IF->SIGMA_SUN;

		// �����������
		gpuOrbitPoint *dpointer_OP;
		cutilSafeCall( hipMalloc( (void**)&dpointer_OP, sizeof( gpuOrbitPoint ) ) );
		cutilSafeCall( hipMemcpy( dpointer_OP, &d_OP, sizeof( gpuOrbitPoint ), hipMemcpyHostToDevice) );

		// buffer constant
		const double TimeStep = 45.0;		// ��� ������������
		const int MaxSizePoints = 3000;		// ������������ ����� ����� �� ������
		const int MaxMinDist = 3000;		// ������������ ����� ������� ���������

		// �������� ������
		double dtt = t - t_s;
		// ����� ����� �� ��������
		int ntt = (int)(dtt/TimeStep);
		printf( "%f %d %f %f\n", dtt, ntt, t_s, t );
		// ������ � ������������ �������
		double *TPR = new double[ntt+1];
		for( int it = 0; it < ntt; it++ )
			TPR[it] = t_s + (it+1)*TimeStep;
		TPR[ntt] = t;

		// init orbit arr
		OrbitArrayPointsGPU LAD;
		LAD.InitArrayList( CU_BlockXYZ, MaxSizePoints );
		
		OrbitArrayPointsCPU LAC;
		LAC.InitArrayList( CU_BlockXYZ, MaxSizePoints );
		
		// ���������
		double *ResultFindMinDist = new double[3*MaxMinDist+1]; 
		ResultFindMinDist[0] = 0;

		hipEventRecord ( stop, 0 );
		hipEventSynchronize ( stop );
		hipEventElapsedTime ( &gpuTime, start, stop );
		printf("GPU prepare Time =  %.2f ms\n", gpuTime );
		//---------------------------------------------//

		int maxNp1 = 0;

		// ���������� ���������� � ���������� �����
		kernalOrbitInitParam<<< gridRK, threadsRK>>>( dpointer_OP, h );
		kernalGetStartPoint<<< gridRK, threadsRK>>>( dpointer_OP, h );
		for( int it = 0; it <= ntt; it++ )
		{
			hipEventRecord ( start, 0 );
			kernalOrbitPredict<<< gridRK, threadsRK>>>( dpointer_OP, h, TPR[it], e, LAD.d_array_list, true );
			LAD.CopyFromGPU( LAC );
			hipDeviceSynchronize();
			hipEventRecord ( stop, 0 );
			hipEventSynchronize ( stop );
			hipEventElapsedTime ( &gpuTime, start, stop );
			printf("GPU cuGetStartPoint and OrbitPredict Time =  %.2f ms\n", gpuTime );

			int dt1 = clock();
			for( int it = 1; it < CU_BlockXYZ; it++ )
			{
				int res = FindSmallDistant( LAC.array_list[0], LAC.array_list[it], it, ResultFindMinDist );
				if( res > maxNp1 )
					maxNp1 = res;
				printf( "%d\r", maxNp1 );
			}
			int dt2 = clock();
			printf("TIME ALL GPU Find Close Approach %f  ms\n", (double)(dt2-dt1)/CLOCKS_PER_SEC*1000.0 );
		}

		FILE *Fres = fopen( "DistInterpolate_gpul.txt", "w" );
		int nf = ResultFindMinDist[0];
		for( int it = 0; it < nf; it++ )
		{
			int index = 3*it + 1;
			int iarr = ResultFindMinDist[index];
			int idn = ListSat.GetSatelliteID( iarr );
			fprintf( Fres, "%d\t %d\t %f\t %f\n", iarr, idn, ResultFindMinDist[index+1], ResultFindMinDist[index+2] );  
		}
		fclose( Fres );

		// From GPU
		d_OP.CopyFromGPU( ListSatResGpu );
		d_OP.DeleteMemory();
		LAD.FreeArrayList();
		LAC.FreeArrayList();
		delete ResultFindMinDist;
		delete TPR;
*/
};
#endif
//#####################################################################//
